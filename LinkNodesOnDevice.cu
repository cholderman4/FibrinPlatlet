#include "hip/hip_runtime.h"

#include "LinkNodesOnDevice.h"
#include "NodeSystemDevice.h"


void LinkNodesOnDevice(
	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

		//Default fill values at 0.
		thrust::fill(nodeInfoVecs.linksThreadMade.begin(),
			nodeInfoVecs.linksThreadMade.end(), 0);
		
		thrust::fill(nodeInfoVecs.delinksThreadMade.begin(),
			nodeInfoVecs.delinksThreadMade.end(), 0);

		thrust::fill(nodeInfoVecs.idMadeTempLeft.begin(),
				nodeInfoVecs.idMadeTempLeft.end(), 0);

		thrust::fill(nodeInfoVecs.idMadeTempRight.begin(),
				nodeInfoVecs.idMadeTempRight.end(), 0);


		//unsigned globalcount = thrust::count_if(wlcInfoVecs.globalNeighbors.begin(),wlcInfoVecs.globalNeighbors.end(),is_less_than(generalParams.maxNodeCount));
		//std::cout<<"currentEdgeCount varpre: "<< generalParams.currentEdgeCount << std::endl;
		//std::cout<<"currentEdgeCount globalpre: "<< globalcount/2 << std::endl;

		thrust::counting_iterator<unsigned> counter(0);
		thrust::transform(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						counter,
						auxVecs.id_bucket.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						counter,
						auxVecs.id_bucket.begin())) + generalParams.maxNodeCount,
				nodeInfoVecs.linksThreadMade.begin(),//output
				LinkNodesFunctor(
					thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
					thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
					thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
					thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
					thrust::raw_pointer_cast(wlcInfoVecs.currentNodeEdgeCountVector.data()),
					thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()),

					thrust::raw_pointer_cast(auxVecs.id_value_expanded.data()),
					thrust::raw_pointer_cast(auxVecs.keyBegin.data()),
					thrust::raw_pointer_cast(auxVecs.keyEnd.data()),

					generalParams.fiberDiameter,
					generalParams.maxNeighborCount,
					generalParams.maxNodeCount,

					generalParams.maxLinksPerIteration,
					thrust::raw_pointer_cast(nodeInfoVecs.idMadeTempLeft.data()),
					thrust::raw_pointer_cast(nodeInfoVecs.idMadeTempRight.data()) ) );



		/*	for (unsigned i = 0; i < idMadeTempLeft.size(); i++) {
				unsigned varL = idMadeTempLeft[i];
				unsigned varR = idMadeTempRight[i];

				if ((varL != 0) || (varR != 0))
					std::cout<< varL << " " <<varR << std::endl;
			}
		unsigned begin = 479 * generalParams.maxNeighborCount;
		unsigned end = begin + generalParams.maxNeighborCount;
		for (unsigned i = begin; i < end; i++){
			unsigned id = wlcInfoVecs.globalNeighbors[i];
			if (id < generalParams.maxNodeCount){
				std::cout<<" 479: "<< id <<std::endl;
			}
		}
		begin = 1004 * generalParams.maxNeighborCount;
		end = begin + generalParams.maxNeighborCount;
		for (unsigned i = begin; i < end; i++){
			unsigned id = wlcInfoVecs.globalNeighbors[i];
			if (id < generalParams.maxNodeCount){
				std::cout<<" 1004: "<< id <<std::endl;
			}
		}*/
	/*	thrust::counting_iterator<unsigned> counterDeLink(0);

		thrust::transform(
						counterDeLink,
						counterDeLink + generalParams.maxNodeCount,
				nodeInfoVecs.delinksThreadMade.begin(),
				DeLinkCopiesFunctor(
					thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
					thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()),
					thrust::raw_pointer_cast(wlcInfoVecs.currentNodeEdgeCountVector.data()),
					generalParams.maxNeighborCount,
					generalParams.maxNodeCount ) );
*/
		//sort increasing
		//issue here
		thrust::sort_by_key(
			nodeInfoVecs.idMadeTempLeft.begin(),nodeInfoVecs.idMadeTempLeft.end(),
			nodeInfoVecs.idMadeTempRight.begin(),thrust::greater<unsigned>() );

		thrust::stable_sort_by_key(
			nodeInfoVecs.idMadeTempRight.begin(),nodeInfoVecs.idMadeTempRight.end(),
			nodeInfoVecs.idMadeTempLeft.begin(), thrust::greater<unsigned>() );


		unsigned idL_init = nodeInfoVecs.idMadeTempLeft[0];
		unsigned idR_init = nodeInfoVecs.idMadeTempRight[0];

		unsigned count = 0;
		for (unsigned i = 1; i < nodeInfoVecs.idMadeTempLeft.size(); i++) {
			//add extra edges and preferred lengths. Notice the lower and upper must be added since each imparts force to one single node and
			//not the neighboring node to the edge. This is b/c edges are solved per node and not per edge
			unsigned idL = nodeInfoVecs.idMadeTempLeft[i];
			unsigned idR = nodeInfoVecs.idMadeTempRight[i];

			if ((idL == idL_init) && (idR == idR_init)){
				count +=1;
			}
			else {
				count = 0;
			}
			//reset initial id's
			idL_init = idL;
			idR_init = idR;


			if ( ((idL != 0) || (idR != 0) ) && (count == 1)) {

				//count edges
				//std::cout<<"placing id: "<< idL<<" " << idR<<std::endl;

				
				nodeInfoVecs.deviceEdgeLeft[generalParams.currentEdgeCount] = (idL);
				nodeInfoVecs.deviceEdgeRight[generalParams.currentEdgeCount] = (idR);
				generalParams.currentEdgeCount += 1;
			} 

		} 
 

	/*	unsigned globalcount = thrust::count_if(wlcInfoVecs.globalNeighbors.begin(), wlcInfoVecs.globalNeighbors.end(), is_less_than(generalParams.maxNodeCount));

		unsigned linksmade = *(thrust::max_element(linksThreadMade.begin(), linksThreadMade.end() ));
		unsigned delinksmade = *(thrust::max_element(delinksThreadMade.begin(), delinksThreadMade.end() ));
		std::cout<<"max links made this iteration: "<< linksmade << std::endl;
		std::cout<<"max unlinks made this iteration: "<< delinksmade << std::endl;

		std::cout<<"currentEdgeCount var: "<< generalParams.currentEdgeCount << std::endl;
		std::cout<<"currentEdgeCount global "<< globalcount/2 << std::endl;

		unsigned temp= thrust::reduce(	wlcInfoVecs.currentNodeEdgeCountVector.begin(),
			wlcInfoVecs.currentNodeEdgeCountVector.end());
		std::cout<<"currentEdgeCount dev: "<< temp << std::endl;
	*/




};
