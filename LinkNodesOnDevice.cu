
#include "LinkNodesOnDevice.h"
#include "NodeSystemDevice.h"


void LinkNodesOnDevice(
	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {
 
 
		thrust::for_each(  
				thrust::make_zip_iterator(
					thrust::make_tuple(
						auxVecs.bucketKeys.begin(), 
						auxVecs.bucketValues.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						auxVecs.bucketKeys.begin(),
						auxVecs.bucketValues.begin())) + generalParams.maxNodeCount,
				
				LinkNodesFunctor(
					thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
					thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
					thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
					thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
					thrust::raw_pointer_cast(wlcInfoVecs.currentNodeEdgeCountVector.data()),
					thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()),
					thrust::raw_pointer_cast(auxVecs.bucketKeys.data()),
					thrust::raw_pointer_cast(auxVecs.bucketValues.data()),
					thrust::raw_pointer_cast(auxVecs.bucketValuesIncludingNeighbor.data()),
					thrust::raw_pointer_cast(auxVecs.keyBegin.data()),
					thrust::raw_pointer_cast(auxVecs.keyEnd.data()), 
					generalParams.fiberDiameter,
					generalParams.maxNeighborCount,
					generalParams.maxNodeCount, 
					generalParams.maxLinksPerIteration, 
					thrust::raw_pointer_cast(nodeInfoVecs.idEdgesMadeTemp.data()) ) );
		

		unsigned numAdded = thrust::count_if( nodeInfoVecs.idEdgesMadeTemp.begin(), nodeInfoVecs.idEdgesMadeTemp.end(), isNotEqualToZero() );
		if (numAdded != 0) {
			//be aware that if an edge was made, it will appear twice, but if it was made once and removed, it will appear once.
			//sort in increasing order. Then when we hit zero, we are done.  
			thrust::sort(nodeInfoVecs.idEdgesMadeTemp.begin(), nodeInfoVecs.idEdgesMadeTemp.end(),thrust::greater<unsigned>() ); 
			
			unsigned idLast = nodeInfoVecs.idEdgesMadeTemp[0];
			//std::cout<<"numadded "<< numAdded<<std::endl;
			//std::cout<<"idlast "<< idLast<<std::endl;
			//std::cout<<"idnext "<< nodeInfoVecs.idEdgesMadeTemp[1] <<std::endl;
			
			unsigned count = 0;
			for (unsigned i = 1; i<nodeInfoVecs.idEdgesMadeTemp.size(); i++) {
				//add extra edges and preferred lengths. Notice the lower and upper must be added since each imparts force to one single node and 
				//not the neighboring node to the edge. This is b/c edges are solved per node and not per edge
				unsigned id = nodeInfoVecs.idEdgesMadeTemp[i];
				
				if (id == 0){
					break; 
				}
	
				if (id == idLast) { 
					//then id has shown up twice and can be added
					count +=1; 
				}  
				else { 
					count = 0;
				}
	
				if ((id != 0) && (count > 0) ) {
					//count edges 
					//std::cout<<"placing id: "<< id<<std::endl;
				
					//idEdgesMadeHost contains id's in matrix format
					nodeInfoVecs.idEdgesMadeHost.push_back(id);
					unsigned idLeft = id % generalParams.maxNodeCount;
					unsigned idRight = id / generalParams.maxNodeCount;
					nodeInfoVecs.deviceEdgeLeft[generalParams.currentEdgeCount] = (idLeft);
					nodeInfoVecs.deviceEdgeRight[generalParams.currentEdgeCount] = (idRight);
					generalParams.currentEdgeCount += 1;
				} 
				
				idLast = id;//set last id to current
			} 
			
		}																					
		

		thrust::fill(nodeInfoVecs.idEdgesMadeTemp.begin(), 
				nodeInfoVecs.idEdgesMadeTemp.end(), 0);
		
};
