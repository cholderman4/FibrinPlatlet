#include "hip/hip_runtime.h"

#include "LinkNodesOnDevice.h"
#include "NodeSystemDevice.h"


void LinkNodesOnDevice(
	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

		thrust::device_vector<unsigned> linksThreadMade;
		linksThreadMade.resize(generalParams.maxNodeCount);
		thrust::fill(linksThreadMade.begin(),linksThreadMade.end(), 0);
		thrust::device_vector<unsigned> delinksThreadMade;
		delinksThreadMade.resize(generalParams.maxNodeCount);
		thrust::fill(delinksThreadMade.begin(),delinksThreadMade.end(), 0);

		thrust::fill(nodeInfoVecs.idEdgesMadeTemp.begin(),
				nodeInfoVecs.idEdgesMadeTemp.end(), 0);

		thrust::device_vector<unsigned> id;
		thrust::device_vector<unsigned> idMadeTempLeft;
		thrust::device_vector<unsigned> idMadeTempRight;
		id.resize(generalParams.maxNodeCount * generalParams.maxLinksPerIteration);
		idMadeTempLeft.resize(generalParams.maxNodeCount * generalParams.maxLinksPerIteration);
		idMadeTempRight.resize(generalParams.maxNodeCount * generalParams.maxLinksPerIteration);

		thrust::fill(id.begin(),
				id.end(), 0);
		thrust::fill(idMadeTempLeft.begin(),
				idMadeTempLeft.end(), 0);
		thrust::fill(idMadeTempRight.begin(),
				idMadeTempRight.end(), 0);


		unsigned globalcount = thrust::count_if(wlcInfoVecs.globalNeighbors.begin(),wlcInfoVecs.globalNeighbors.end(),is_less_than(generalParams.maxNodeCount));
	//	std::cout<<"currentEdgeCount varpre: "<< generalParams.currentEdgeCount << std::endl;
	//	std::cout<<"currentEdgeCount globalpre: "<< globalcount/2 << std::endl;

		thrust::counting_iterator<unsigned> counter(0);
		thrust::transform(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						counter,
						auxVecs.id_bucket.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						counter,
						auxVecs.id_bucket.begin())) + generalParams.maxNodeCount,
				linksThreadMade.begin(),//output
				LinkNodesFunctor(
					thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
					thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
					thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
					thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
					thrust::raw_pointer_cast(wlcInfoVecs.currentNodeEdgeCountVector.data()),
					thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()),

					thrust::raw_pointer_cast(auxVecs.id_value_expanded.data()),
					thrust::raw_pointer_cast(auxVecs.keyBegin.data()),
					thrust::raw_pointer_cast(auxVecs.keyEnd.data()),

					generalParams.fiberDiameter,
					generalParams.maxNeighborCount,
					generalParams.maxNodeCount,

					generalParams.maxLinksPerIteration,
					thrust::raw_pointer_cast(idMadeTempLeft.data()),
					thrust::raw_pointer_cast(idMadeTempRight.data()) ) );

			thrust::sort_by_key(
				idMadeTempLeft.begin(),idMadeTempLeft.end(),
				idMadeTempRight.begin(),thrust::greater<unsigned>() );

			thrust::stable_sort_by_key(
				idMadeTempRight.begin(),idMadeTempRight.end(),
				idMadeTempLeft.begin(), thrust::greater<unsigned>() );


		/*	for (unsigned i = 0; i < idMadeTempLeft.size(); i++) {
				unsigned varL = idMadeTempLeft[i];
				unsigned varR = idMadeTempRight[i];

				if ((varL != 0) || (varR != 0))
					std::cout<< varL << " " <<varR << std::endl;
			}*/
	/*	unsigned begin = 479 * generalParams.maxNeighborCount;
		unsigned end = begin + generalParams.maxNeighborCount;
		for (unsigned i = begin; i < end; i++){
			unsigned id = wlcInfoVecs.globalNeighbors[i];
			if (id < generalParams.maxNodeCount){
				std::cout<<" 479: "<< id <<std::endl;
			}
		}
		begin = 1004 * generalParams.maxNeighborCount;
		end = begin + generalParams.maxNeighborCount;
		for (unsigned i = begin; i < end; i++){
			unsigned id = wlcInfoVecs.globalNeighbors[i];
			if (id < generalParams.maxNodeCount){
				std::cout<<" 1004: "<< id <<std::endl;
			}
		}*/
		thrust::counting_iterator<unsigned> counterDeLink(0);

		thrust::transform(
						counterDeLink,
						counterDeLink + generalParams.maxNodeCount,
				delinksThreadMade.begin(),
				DeLinkCopiesFunctor(
					thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
					thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()),
					thrust::raw_pointer_cast(wlcInfoVecs.currentNodeEdgeCountVector.data()),
					generalParams.maxNeighborCount,
					generalParams.maxNodeCount ) );

    	unsigned endKey = thrust::get<0>(
    	    thrust::unique_by_key(
    	        idMadeTempRight.begin(), idMadeTempRight.end(),
				idMadeTempLeft.begin(),
    	    thrust::equal_to<unsigned>() )) - idMadeTempRight.begin();//binary_pred


		for (unsigned i = 0; i < endKey; i++) {
			//add extra edges and preferred lengths. Notice the lower and upper must be added since each imparts force to one single node and
			//not the neighboring node to the edge. This is b/c edges are solved per node and not per edge
			unsigned idL = idMadeTempLeft[i];
			unsigned idR = idMadeTempRight[i];


			if ((idL != 0) || (idR != 0) ) {
				//count edges
				//std::cout<<"placing id: "<< idL<<" " << idR<<std::endl;

				//idEdgesMadeHost contains id's in matrix format
				nodeInfoVecs.deviceEdgeLeft[generalParams.currentEdgeCount] = (idL);
				nodeInfoVecs.deviceEdgeRight[generalParams.currentEdgeCount] = (idR);
				generalParams.currentEdgeCount += 1;
			} 

		} 


		globalcount = thrust::count_if(wlcInfoVecs.globalNeighbors.begin(), wlcInfoVecs.globalNeighbors.end(), is_less_than(generalParams.maxNodeCount));

		unsigned linksmade = *(thrust::max_element(linksThreadMade.begin(), linksThreadMade.end() ));
		unsigned delinksmade = *(thrust::max_element(delinksThreadMade.begin(), delinksThreadMade.end() ));
	/*	std::cout<<"max links made this iteration: "<< linksmade << std::endl;
		std::cout<<"max unlinks made this iteration: "<< delinksmade << std::endl;

		std::cout<<"currentEdgeCount var: "<< generalParams.currentEdgeCount << std::endl;
		std::cout<<"currentEdgeCount global "<< globalcount/2 << std::endl;
*/




};
