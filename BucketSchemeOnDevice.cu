#include "hip/hip_runtime.h"

#include "BucketSchemeOnDevice.h"
#include "NodeSystemDevice.h"

//take domain and discretize into square buckets of size gridspace
void initDimensionBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	double minXTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocX.begin(), nodeInfoVecs.nodeLocX.end())));
	double maxXTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocX.begin(), nodeInfoVecs.nodeLocX.end())));
	double minYTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocY.begin(), nodeInfoVecs.nodeLocY.end())));
	double maxYTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocY.begin(), nodeInfoVecs.nodeLocY.end())));
	double minZTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocZ.begin(), nodeInfoVecs.nodeLocZ.end())));
	double maxZTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocZ.begin(), nodeInfoVecs.nodeLocZ.end())));

	//platelets
	domainParams.pltminX = (*(thrust::min_element(pltInfoVecs.pltLocX.begin(), pltInfoVecs.pltLocX.end())));
	domainParams.pltmaxX = (*(thrust::max_element(pltInfoVecs.pltLocX.begin(), pltInfoVecs.pltLocX.end())));
	domainParams.pltminY = (*(thrust::min_element(pltInfoVecs.pltLocY.begin(), pltInfoVecs.pltLocY.end())));
	domainParams.pltmaxY = (*(thrust::max_element(pltInfoVecs.pltLocY.begin(), pltInfoVecs.pltLocY.end())));
	domainParams.pltminZ = (*(thrust::min_element(pltInfoVecs.pltLocZ.begin(), pltInfoVecs.pltLocZ.end())));
	domainParams.pltmaxZ = (*(thrust::max_element(pltInfoVecs.pltLocZ.begin(), pltInfoVecs.pltLocZ.end())));

	double space = 0.0;
	domainParams.minX = min(minXTemp, domainParams.pltminX) - space;
	domainParams.maxX = max(maxXTemp, domainParams.pltmaxX) + space;
	domainParams.minY = min(minYTemp, domainParams.pltminY) - space;
	domainParams.maxY = max(maxYTemp, domainParams.pltmaxY) + space;
	domainParams.minZ = min(minZTemp, domainParams.pltminZ) - space;
	domainParams.maxZ = max(maxZTemp, domainParams.pltmaxZ) + space;

	domainParams.XBucketCount = (ceil(domainParams.maxX - domainParams.minX) / domainParams.gridSpacing) + 1;
	domainParams.YBucketCount = (ceil(domainParams.maxY - domainParams.minY) / domainParams.gridSpacing) + 1;
	domainParams.ZBucketCount = (ceil(domainParams.maxZ - domainParams.minZ) / domainParams.gridSpacing) + 1;
	
	if ( (domainParams.XBucketCount * domainParams.YBucketCount * domainParams.ZBucketCount) != domainParams.totalBucketCount	) {

		//double amount of buckets in case of resizing networks
		domainParams.totalBucketCount = domainParams.XBucketCount * domainParams.YBucketCount * domainParams.ZBucketCount;
		std::cout<<"grid: "<< domainParams.gridSpacing << std::endl;
		std::cout<<"total bucket count: "<< domainParams.totalBucketCount<<std::endl;

		auxVecs.keyBegin.resize(domainParams.totalBucketCount);
		auxVecs.keyEnd.resize(domainParams.totalBucketCount);
		//platelets
		auxVecs.keyPltBegin.resize(domainParams.totalBucketCount);
		auxVecs.keyPltEnd.resize(domainParams.totalBucketCount);

	}

	thrust::fill(auxVecs.keyBegin.begin(),auxVecs.keyBegin.end(),0);
	thrust::fill(auxVecs.keyEnd.begin(),auxVecs.keyEnd.end(),0);
	//platelets
	thrust::fill(auxVecs.keyPltBegin.begin(),auxVecs.keyPltBegin.end(),0);
	thrust::fill(auxVecs.keyPltEnd.begin(),auxVecs.keyPltEnd.end(),0);

};

//convert buckets into neighboring scheme
void extendBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs) {

	//memory is already allocated.
	unsigned endIndexExpanded = (auxVecs.endIndexBucketKeys) * 27;
	//platelets
	unsigned endIndexPltExpanded = (auxVecs.endIndexBucketPltKeys) * 27;

	//test for removing copies.
	unsigned valuesCount = auxVecs.bucketValues.size();
	thrust::fill(auxVecs.bucketKeysExpanded.begin(),auxVecs.bucketKeysExpanded.end(),0);
	thrust::fill(auxVecs.bucketValuesIncludingNeighbor.begin(),auxVecs.bucketValuesIncludingNeighbor.end(),0);

	thrust::fill(auxVecs.bucketPltKeysExpanded.begin(),auxVecs.bucketPltKeysExpanded.end(),0);
	thrust::fill(auxVecs.bucketPltValuesIncludingNeighbor.begin(),auxVecs.bucketPltValuesIncludingNeighbor.end(),0);




	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> first(27);
	/**
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/
	thrust::constant_iterator<unsigned> last = first + (auxVecs.endIndexBucketKeys); // this is NOT numerical addition!

	expand(first, last,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketKeys.begin(),
				auxVecs.bucketValues.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketKeysExpanded.begin(),
				auxVecs.bucketValuesIncludingNeighbor.begin())));


	thrust::counting_iterator<unsigned> countingBegin(0);

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketKeysExpanded.begin(),
				countingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketKeysExpanded.begin(),
				countingBegin)) + endIndexExpanded,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketKeysExpanded.begin(),
				countingBegin)),
		NeighborFunctor(
			domainParams.XBucketCount,
			domainParams.YBucketCount,
			domainParams.ZBucketCount));



	unsigned numberOfOutOfRange = thrust::count_if(auxVecs.bucketKeysExpanded.begin(),
		auxVecs.bucketKeysExpanded.end(), is_greater_than(domainParams.totalBucketCount) );
	unsigned numberInsideRange = endIndexExpanded - numberOfOutOfRange;

	//unsigned endIndexSearch = endIndexExpanded - numberOfOutOfRange;

	thrust::stable_sort_by_key(auxVecs.bucketKeysExpanded.begin(),
		auxVecs.bucketKeysExpanded.begin() + endIndexExpanded,
		auxVecs.bucketValuesIncludingNeighbor.begin());
	
	numberInsideRange = 
		thrust::get<0>(thrust::unique_by_key(auxVecs.bucketValuesIncludingNeighbor.begin(),
			auxVecs.bucketValuesIncludingNeighbor.begin() + endIndexExpanded,
			auxVecs.bucketKeysExpanded.begin())) - auxVecs.bucketValuesIncludingNeighbor.begin();

	auxVecs.bucketKeysExpanded.erase(
			auxVecs.bucketKeysExpanded.begin() + numberInsideRange,
			auxVecs.bucketKeysExpanded.end());

	auxVecs.bucketValuesIncludingNeighbor.erase(
			auxVecs.bucketValuesIncludingNeighbor.begin() + numberInsideRange,
			auxVecs.bucketValuesIncludingNeighbor.end());




	thrust::counting_iterator<unsigned> search_begin(0);

	thrust::lower_bound(auxVecs.bucketKeysExpanded.begin(),
		auxVecs.bucketKeysExpanded.end(), search_begin,
		search_begin + domainParams.totalBucketCount,
		auxVecs.keyBegin.begin());

	thrust::upper_bound(auxVecs.bucketKeysExpanded.begin(),
		auxVecs.bucketKeysExpanded.end(),search_begin,
		search_begin + domainParams.totalBucketCount,
		auxVecs.keyEnd.begin());

	//platelets
	/*unsigned valuesPltCount = auxVecs.bucketPltValues.size();
	thrust::fill(auxVecs.bucketPltKeysExpanded.begin(),auxVecs.bucketPltKeysExpanded.end(),0);
	thrust::fill(auxVecs.bucketPltValuesIncludingNeighbor.begin(),auxVecs.bucketPltValuesIncludingNeighbor.end(),0);
*/



	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> pltfirst(27);
	/**
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/
	
	
	thrust::constant_iterator<unsigned> pltlast = pltfirst + (auxVecs.endIndexBucketPltKeys); // this is NOT numerical addition!

	expand(pltfirst, pltlast,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketPltKeys.begin(),
				auxVecs.bucketPltValues.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketPltKeysExpanded.begin(),
				auxVecs.bucketPltValuesIncludingNeighbor.begin())));


	thrust::counting_iterator<unsigned> pltcountingBegin(0);

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketPltKeysExpanded.begin(),
				pltcountingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketPltKeysExpanded.begin(),
				pltcountingBegin)) + endIndexPltExpanded,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketPltKeysExpanded.begin(),
				pltcountingBegin)),
		NeighborFunctor(
			domainParams.XBucketCount,
			domainParams.YBucketCount,
			domainParams.ZBucketCount));



	unsigned pltnumberOfOutOfRange = thrust::count_if(auxVecs.bucketPltKeysExpanded.begin(),
		auxVecs.bucketPltKeysExpanded.end(), is_greater_than(domainParams.totalBucketCount) );
	unsigned pltnumberInsideRange = endIndexPltExpanded - pltnumberOfOutOfRange;

	//unsigned endIndexPltSearch = endIndexPltExpanded - pltnumberOfOutOfRange;

	thrust::sort_by_key(auxVecs.bucketPltKeysExpanded.begin(),
		auxVecs.bucketPltKeysExpanded.begin() + endIndexPltExpanded,
		auxVecs.bucketPltValuesIncludingNeighbor.begin());
	
	pltnumberInsideRange = 
		thrust::get<0>(thrust::unique_by_key(auxVecs.bucketPltValuesIncludingNeighbor.begin(),
			auxVecs.bucketPltValuesIncludingNeighbor.begin() + endIndexExpanded,
			auxVecs.bucketPltKeysExpanded.begin())) - auxVecs.bucketPltValuesIncludingNeighbor.begin();

	auxVecs.bucketPltKeysExpanded.erase(
			auxVecs.bucketPltKeysExpanded.begin() + pltnumberInsideRange,
			auxVecs.bucketPltKeysExpanded.end());

	auxVecs.bucketPltValuesIncludingNeighbor.erase(
			auxVecs.bucketPltValuesIncludingNeighbor.begin() + pltnumberInsideRange,
			auxVecs.bucketPltValuesIncludingNeighbor.end());




	thrust::counting_iterator<unsigned> pltsearch_begin(0);

	thrust::lower_bound(auxVecs.bucketPltKeysExpanded.begin(),
		auxVecs.bucketPltKeysExpanded.end(), pltsearch_begin,
		pltsearch_begin + domainParams.totalBucketCount,
		auxVecs.keyPltBegin.begin());

	thrust::upper_bound(auxVecs.bucketPltKeysExpanded.begin(),
		auxVecs.bucketPltKeysExpanded.end(),pltsearch_begin,
		pltsearch_begin + domainParams.totalBucketCount,
		auxVecs.keyPltEnd.begin());

}


//takes nodes and places in buckets.
void buildBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {


	thrust::counting_iterator<unsigned> indexBucketBegin(0);
	// takes counting iterator and coordinates
	// return tuple of keys and values
	// transform the points to their bucket indices
	
	//std::cout<<"bucket nodes"<<std::endl;
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)) + generalParams.maxNodeCount,
		BucketIndexer(
			domainParams.minX, domainParams.maxX, domainParams.minY,
			domainParams.maxY, domainParams.minZ, domainParams.maxZ,
			domainParams.gridSpacing,
			thrust::raw_pointer_cast(auxVecs.bucketKeys.data()),
			thrust::raw_pointer_cast(auxVecs.bucketValues.data())));

//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.bucketValues.begin(),
		auxVecs.bucketValues.begin() + generalParams.maxNodeCount,
		auxVecs.bucketKeys.begin());
unsigned numberOutOfRange = thrust::count(auxVecs.bucketKeys.begin(),
			auxVecs.bucketKeys.begin() + generalParams.maxNodeCount, ULONG_MAX);

	auxVecs.endIndexBucketKeys = generalParams.maxNodeCount - numberOutOfRange;

	//platelets
	//std::cout<<"bucket platelet"<<std::endl;
	thrust::counting_iterator<unsigned> indexBucketBegin1(0);
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				pltInfoVecs.pltLocX.begin(),
				pltInfoVecs.pltLocY.begin(),
				pltInfoVecs.pltLocZ.begin(),
				indexBucketBegin1)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				pltInfoVecs.pltLocX.begin(),
				pltInfoVecs.pltLocY.begin(),
				pltInfoVecs.pltLocZ.begin(),
				indexBucketBegin1)) + generalParams.maxPltCount,
		BucketIndexer(
			domainParams.minX, domainParams.maxX, domainParams.minY,
			domainParams.maxY, domainParams.minZ, domainParams.maxZ,
			domainParams.gridSpacing,
			thrust::raw_pointer_cast(auxVecs.bucketPltKeys.data()),
			thrust::raw_pointer_cast(auxVecs.bucketPltValues.data())));

			
	//std::cout<<"end bucket platelet"<<std::endl;
//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.bucketPltValues.begin(),
		auxVecs.bucketPltValues.begin() + generalParams.maxPltCount,
		auxVecs.bucketPltKeys.begin());
	
unsigned numberPltOutOfRange = thrust::count(auxVecs.bucketPltKeys.begin(),
			auxVecs.bucketPltKeys.begin() + generalParams.maxPltCount, ULONG_MAX);

	auxVecs.endIndexBucketPltKeys = generalParams.maxPltCount - numberPltOutOfRange;


};
