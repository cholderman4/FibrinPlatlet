#include "hip/hip_runtime.h"

#include "BucketSchemeOnDevice.h"
#include "NodeSystemDevice.h"

//take domain and discretize into square buckets of size gridspace
void initDimensionBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	double minXTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocX.begin(), nodeInfoVecs.nodeLocX.end())));
	double maxXTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocX.begin(), nodeInfoVecs.nodeLocX.end())));
	double minYTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocY.begin(), nodeInfoVecs.nodeLocY.end())));
	double maxYTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocY.begin(), nodeInfoVecs.nodeLocY.end())));
	double minZTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocZ.begin(), nodeInfoVecs.nodeLocZ.end())));
	double maxZTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocZ.begin(), nodeInfoVecs.nodeLocZ.end())));

	//platelets
	domainParams.pltminX = (*(thrust::min_element(pltInfoVecs.pltLocX.begin(), pltInfoVecs.pltLocX.end())));
	domainParams.pltmaxX = (*(thrust::max_element(pltInfoVecs.pltLocX.begin(), pltInfoVecs.pltLocX.end())));
	domainParams.pltminY = (*(thrust::min_element(pltInfoVecs.pltLocY.begin(), pltInfoVecs.pltLocY.end())));
	domainParams.pltmaxY = (*(thrust::max_element(pltInfoVecs.pltLocY.begin(), pltInfoVecs.pltLocY.end())));
	domainParams.pltminZ = (*(thrust::min_element(pltInfoVecs.pltLocZ.begin(), pltInfoVecs.pltLocZ.end())));
	domainParams.pltmaxZ = (*(thrust::max_element(pltInfoVecs.pltLocZ.begin(), pltInfoVecs.pltLocZ.end())));

	double space = 0.0;
	domainParams.minX = min(minXTemp, domainParams.pltminX) - space;
	domainParams.maxX = max(maxXTemp, domainParams.pltmaxX) + space;
	domainParams.minY = min(minYTemp, domainParams.pltminY) - space;
	domainParams.maxY = max(maxYTemp, domainParams.pltmaxY) + space;
	domainParams.minZ = min(minZTemp, domainParams.pltminZ) - space;
	domainParams.maxZ = max(maxZTemp, domainParams.pltmaxZ) + space;

	//always set bucket count. Update total if different. 
	domainParams.XBucketCount = ceil((domainParams.maxX - domainParams.minX) / domainParams.gridSpacing + 1);
	domainParams.YBucketCount = ceil((domainParams.maxY - domainParams.minY) / domainParams.gridSpacing + 1);
	domainParams.ZBucketCount = ceil((domainParams.maxZ - domainParams.minZ) / domainParams.gridSpacing + 1);

	if ( (domainParams.XBucketCount * domainParams.YBucketCount * domainParams.ZBucketCount) != domainParams.totalBucketCount	) {
		std::cout<<"x-bucket: "<< domainParams.XBucketCount<<std::endl;
		std::cout<<"y-bucket: "<< domainParams.YBucketCount<<std::endl;
		std::cout<<"z-bucket: "<< domainParams.ZBucketCount<<std::endl;
		//double amount of buckets in case of resizing networks
		domainParams.totalBucketCount = domainParams.XBucketCount * domainParams.YBucketCount * domainParams.ZBucketCount;
		std::cout<<"grid: "<< domainParams.gridSpacing << std::endl;
		std::cout<<"total bucket count: "<< domainParams.totalBucketCount<<std::endl;

		auxVecs.keyBegin.resize(domainParams.totalBucketCount);
		auxVecs.keyEnd.resize(domainParams.totalBucketCount);
		//platelets
		auxVecs.keyPltBegin.resize(domainParams.totalBucketCount); 
		auxVecs.keyPltEnd.resize(domainParams.totalBucketCount); 
 
	}

	thrust::fill(auxVecs.keyBegin.begin(),auxVecs.keyBegin.end(),0);
	thrust::fill(auxVecs.keyEnd.begin(),auxVecs.keyEnd.end(),0);
	//platelets
	thrust::fill(auxVecs.keyPltBegin.begin(),auxVecs.keyPltBegin.end(),0);
	thrust::fill(auxVecs.keyPltEnd.begin(),auxVecs.keyPltEnd.end(),0);

};

//convert buckets into neighboring scheme
void extendBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	//memory is already allocated.
	unsigned endIndexExpanded = (auxVecs.endIndexBucketKeys) * 27;
	//platelets
	unsigned endIndexPltExpanded = (auxVecs.endIndexBucketPltKeys) * 27;

	//test for removing copies.
	unsigned valuesCount = auxVecs.id_value.size();
	thrust::fill(auxVecs.id_bucket_expanded.begin(),auxVecs.id_bucket_expanded.end(),0);
	thrust::fill(auxVecs.id_value_expanded.begin(),auxVecs.id_value_expanded.end(),0);

	thrust::fill(auxVecs.idPlt_bucket_expanded.begin(),auxVecs.idPlt_bucket_expanded.end(),0);
	thrust::fill(auxVecs.idPlt_value_expanded.begin(),auxVecs.idPlt_value_expanded.end(),0);




	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> first(27);
	/*
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/
	thrust::constant_iterator<unsigned> last = first + (auxVecs.endIndexBucketKeys); // this is NOT numerical addition!

	expand(first, last,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket.begin(),
				auxVecs.id_value.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded.begin(),
				auxVecs.id_value_expanded.begin())));

	thrust::counting_iterator<unsigned> countingBegin(0);
 
	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded.begin(),
				countingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded.begin(),
				countingBegin)) + endIndexExpanded,
		
		auxVecs.id_bucket_expanded.begin(),
		NeighborFunctor(
			domainParams.XBucketCount,
			domainParams.YBucketCount,
			domainParams.ZBucketCount)); 

	thrust::stable_sort_by_key(auxVecs.id_bucket_expanded.begin(),
		auxVecs.id_bucket_expanded.end(),
		auxVecs.id_value_expanded.begin());


	thrust::counting_iterator<unsigned> search_begin(0);

	thrust::lower_bound(auxVecs.id_bucket_expanded.begin(),
		auxVecs.id_bucket_expanded.end(), search_begin,
		search_begin + domainParams.totalBucketCount,
		auxVecs.keyBegin.begin());

	thrust::upper_bound(auxVecs.id_bucket_expanded.begin(),
		auxVecs.id_bucket_expanded.end(),search_begin,
		search_begin + domainParams.totalBucketCount,
		auxVecs.keyEnd.begin());

	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> pltfirst(27);
	/**
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/


	thrust::constant_iterator<unsigned> pltlast = pltfirst + (auxVecs.endIndexBucketPltKeys); // this is NOT numerical addition!

	expand(pltfirst, pltlast,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket.begin(),
				auxVecs.idPlt_value.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket_expanded.begin(),
				auxVecs.idPlt_value_expanded.begin())));


	thrust::counting_iterator<unsigned> pltcountingBegin(0);

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket_expanded.begin(),
				pltcountingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket_expanded.begin(),
				pltcountingBegin)) + endIndexPltExpanded,
			auxVecs.idPlt_bucket_expanded.begin(),
		NeighborFunctor(
			domainParams.XBucketCount,
			domainParams.YBucketCount,
			domainParams.ZBucketCount));



	//unsigned pltnumberOfOutOfRange = thrust::count_if(auxVecs.idPlt_bucket_expanded.begin(),
	//	auxVecs.idPlt_bucket_expanded.end(), is_greater_than(domainParams.totalBucketCount) );
	//unsigned pltnumberInsideRange = endIndexPltExpanded - pltnumberOfOutOfRange;

	//unsigned endIndexPltSearch = endIndexPltExpanded - pltnumberOfOutOfRange;

	thrust::sort_by_key(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.end(),
		auxVecs.idPlt_value_expanded.begin());
	
	thrust::counting_iterator<unsigned> pltsearch_begin(0);

	thrust::lower_bound(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.end(), pltsearch_begin,
		pltsearch_begin + domainParams.totalBucketCount,
		auxVecs.keyPltBegin.begin());

	thrust::upper_bound(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.end(),pltsearch_begin,
		pltsearch_begin + domainParams.totalBucketCount,
		auxVecs.keyPltEnd.begin());

	
	unsigned choice = 0;

	unsigned bucket = auxVecs.idPlt_bucket[choice];
	std::cout<<"bucketplt 0: "<< bucket<<std::endl;
	unsigned begin = auxVecs.keyBegin[bucket];
	unsigned end = auxVecs.keyEnd[bucket];
	
	std::cout<<"from bucket scheme:"<<std::endl;
	for (unsigned i = begin; i < end; i++) {
		
		unsigned nbr = auxVecs.id_value_expanded[i];
		unsigned buck = auxVecs.id_bucket[nbr];
		double x_dist = pltInfoVecs.pltLocX[choice] - nodeInfoVecs.nodeLocX[nbr];
		double y_dist = pltInfoVecs.pltLocY[choice] - nodeInfoVecs.nodeLocY[nbr];
		double z_dist = pltInfoVecs.pltLocZ[choice] - nodeInfoVecs.nodeLocZ[nbr];
		double dist = std::sqrt(std::pow(x_dist,2.0)+std::pow(y_dist,2.0)+std::pow(z_dist,2.0));
		if (dist < 1.0){
			std::cout<<"dist: "<< dist<< " between: "<< choice << " and nbr: "<< nbr<<std::endl; 
			std::cout<<"nbr: "<< nbr<< " is in bucket: "<< buck <<std::endl;
		}
	}

	/*
	std::cout<<"from all plt:"<<std::endl;
	for (unsigned i = 0; i < generalParams.maxNodeCount; i++) {
		unsigned nbr = i;//auxVecs.id_value_expanded[i];
		unsigned buck = auxVecs.id_bucket[nbr];
		double x_dist = pltInfoVecs.pltLocX[choice] - nodeInfoVecs.nodeLocX[nbr];
		double y_dist = pltInfoVecs.pltLocY[choice] - nodeInfoVecs.nodeLocY[nbr];
		double z_dist = pltInfoVecs.pltLocZ[choice] - nodeInfoVecs.nodeLocZ[nbr];
		double dist = std::sqrt(std::pow(x_dist,2.0)+std::pow(y_dist,2.0)+std::pow(z_dist,2.0));
		if (dist < 1.0){
			std::cout<<"dist: "<< dist<< " between: "<< choice << " and nbr: "<< nbr<<std::endl; 
			std::cout<<"nbr: "<< nbr<< " is in bucket: "<< buck <<std::endl;
		} 
	}*/


}


//takes nodes and places in buckets.
void buildBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {


	thrust::counting_iterator<unsigned> indexBucketBegin(0);
	// takes counting iterator and coordinates
	// return tuple of keys and values
	// transform the points to their bucket indices

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)) + generalParams.maxNodeCount,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket.begin(),
				auxVecs.id_value.begin())),
		BucketIndexer(
			domainParams.minX, domainParams.maxX, domainParams.minY,
			domainParams.maxY, domainParams.minZ, domainParams.maxZ,
			domainParams.XBucketCount,domainParams.YBucketCount,domainParams.ZBucketCount,
			domainParams.gridSpacing));

//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.id_value.begin(),
		auxVecs.id_value.begin() + generalParams.maxNodeCount,
		auxVecs.id_bucket.begin());

	//platelets
	//std::cout<<"bucket platelet"<<std::endl;
	thrust::counting_iterator<unsigned> indexBucketBegin1(0);
	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				pltInfoVecs.pltLocX.begin(),
				pltInfoVecs.pltLocY.begin(),
				pltInfoVecs.pltLocZ.begin(),
				indexBucketBegin1)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				pltInfoVecs.pltLocX.begin(),
				pltInfoVecs.pltLocY.begin(),
				pltInfoVecs.pltLocZ.begin(),
				indexBucketBegin1)) + generalParams.maxPltCount,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket.begin(),
				auxVecs.idPlt_value.begin())),
		BucketIndexer(
			domainParams.minX, domainParams.maxX, domainParams.minY,
			domainParams.maxY, domainParams.minZ, domainParams.maxZ,
			domainParams.XBucketCount,domainParams.YBucketCount,domainParams.ZBucketCount,
			domainParams.gridSpacing));


	//std::cout<<"end bucket platelet"<<std::endl;
//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.idPlt_value.begin(),
		auxVecs.idPlt_value.end(),
		auxVecs.idPlt_bucket.begin());


};
