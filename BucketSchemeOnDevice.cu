#include "hip/hip_runtime.h"

#include "BucketSchemeOnDevice.h"
#include "NodeSystemDevice.h"

//take domain and discretize into square buckets of size gridspace
void initDimensionBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	domainParams.minX = (*(thrust::min_element(nodeInfoVecs.nodeLocX.begin(), nodeInfoVecs.nodeLocX.end())));
	domainParams.maxX = (*(thrust::max_element(nodeInfoVecs.nodeLocX.begin(), nodeInfoVecs.nodeLocX.end())));
	domainParams.minY = (*(thrust::min_element(nodeInfoVecs.nodeLocY.begin(), nodeInfoVecs.nodeLocY.end())));
	domainParams.maxY = (*(thrust::max_element(nodeInfoVecs.nodeLocY.begin(), nodeInfoVecs.nodeLocY.end())));
	domainParams.minZ = (*(thrust::min_element(nodeInfoVecs.nodeLocZ.begin(), nodeInfoVecs.nodeLocZ.end())));
	domainParams.maxZ = (*(thrust::max_element(nodeInfoVecs.nodeLocZ.begin(), nodeInfoVecs.nodeLocZ.end())));

	//platelets
	domainParams.pltminX = (*(thrust::min_element(pltInfoVecs.pltLocX.begin(), pltInfoVecs.pltLocX.end())));
	domainParams.pltmaxX = (*(thrust::max_element(pltInfoVecs.pltLocX.begin(), pltInfoVecs.pltLocX.end())));
	domainParams.pltminY = (*(thrust::min_element(pltInfoVecs.pltLocY.begin(), pltInfoVecs.pltLocY.end())));
	domainParams.pltmaxY = (*(thrust::max_element(pltInfoVecs.pltLocY.begin(), pltInfoVecs.pltLocY.end())));
	domainParams.pltminZ = (*(thrust::min_element(pltInfoVecs.pltLocZ.begin(), pltInfoVecs.pltLocZ.end())));
	domainParams.pltmaxZ = (*(thrust::max_element(pltInfoVecs.pltLocZ.begin(), pltInfoVecs.pltLocZ.end())));

	if (domainParams.minX > domainParams.pltminX){
		domainParams.minX = domainParams.pltminX;
	}
	if (domainParams.maxX < domainParams.pltmaxX){
		domainParams.maxX = domainParams.pltmaxX;
	}
	if (domainParams.minY > domainParams.pltminY){
		domainParams.minY = domainParams.pltminY;
	}
	if (domainParams.maxY < domainParams.pltmaxY){
		domainParams.maxY = domainParams.pltmaxY;
	}
	if (domainParams.minZ > domainParams.pltminZ){
		domainParams.minZ = domainParams.pltminZ;
	}
	if (domainParams.maxZ < domainParams.pltmaxZ){
		domainParams.maxZ = domainParams.pltmaxZ;
	}


	if (generalParams.iterationCounter == 0) {
		domainParams.originMinX = domainParams.minX;
		domainParams.originMaxX = domainParams.maxX;
		domainParams.originMinY = domainParams.minY;
		domainParams.originMaxY = domainParams.maxY;
		domainParams.originMinZ = domainParams.minZ;
		domainParams.originMaxZ = domainParams.maxZ;

		//double amount of buckets in case of resizing networks
		domainParams.XBucketCount = 2.0 * ceil(ceil(domainParams.maxX - domainParams.minX) / domainParams.gridSpacing) + 1;
		domainParams.YBucketCount = 2.0 * ceil(ceil(domainParams.maxY - domainParams.minY) / domainParams.gridSpacing) + 1;
		domainParams.ZBucketCount = 2.0 * ceil(ceil(domainParams.maxZ - domainParams.minZ) / domainParams.gridSpacing) + 1;
		domainParams.totalBucketCount = domainParams.XBucketCount * domainParams.YBucketCount * domainParams.ZBucketCount;

		if (generalParams.iterationCounter == 0 )
			std::cout<<"total bucket count: "<< domainParams.totalBucketCount<<std::endl;

		auxVecs.keyBegin.resize(domainParams.totalBucketCount);
		auxVecs.keyEnd.resize(domainParams.totalBucketCount);
		//platelets
		auxVecs.keyPltBegin.resize(domainParams.totalBucketCount);
		auxVecs.keyPltEnd.resize(domainParams.totalBucketCount);
	}
	thrust::fill(auxVecs.keyBegin.begin(),auxVecs.keyBegin.end(),0);
	thrust::fill(auxVecs.keyEnd.begin(),auxVecs.keyEnd.end(),0);
	//platelets
	thrust::fill(auxVecs.keyPltBegin.begin(),auxVecs.keyPltBegin.end(),0);
	thrust::fill(auxVecs.keyPltEnd.begin(),auxVecs.keyPltEnd.end(),0);

}

//convert buckets into neighboring scheme
void extendBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs) {

	//memory is already allocated.
	unsigned endIndexExpanded = (auxVecs.endIndexBucketKeys) * 27;
	//platelets
	unsigned endIndexPltExpanded = (auxVecs.endIndexBucketPltKeys) * 27;

	//test for removing copies.
	unsigned valuesCount = auxVecs.bucketValues.size();
	thrust::fill(auxVecs.bucketKeysExpanded.begin(),auxVecs.bucketKeysExpanded.end(),0);
	thrust::fill(auxVecs.bucketValuesIncludingNeighbor.begin(),auxVecs.bucketValuesIncludingNeighbor.end(),0);




	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> first(27);
	/**
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/
	thrust::constant_iterator<unsigned> last = first + (auxVecs.endIndexBucketKeys); // this is NOT numerical addition!

	expand(first, last,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketKeys.begin(),
				auxVecs.bucketValues.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketKeysExpanded.begin(),
				auxVecs.bucketValuesIncludingNeighbor.begin())));


	thrust::counting_iterator<unsigned> countingBegin(0);

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketKeysExpanded.begin(),
				countingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketKeysExpanded.begin(),
				countingBegin)) + endIndexExpanded,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketKeysExpanded.begin(),
				countingBegin)),
		NeighborFunctor(
			domainParams.XBucketCount,
			domainParams.YBucketCount,
			domainParams.ZBucketCount));



	unsigned numberOfOutOfRange = thrust::count_if(auxVecs.bucketKeysExpanded.begin(),
		auxVecs.bucketKeysExpanded.end(), is_greater_than(domainParams.totalBucketCount) );
	unsigned numberInsideRange = endIndexExpanded - numberOfOutOfRange;

	//unsigned endIndexSearch = endIndexExpanded - numberOfOutOfRange;

	thrust::sort_by_key(auxVecs.bucketKeysExpanded.begin(),
		auxVecs.bucketKeysExpanded.begin() + endIndexExpanded,
		auxVecs.bucketValuesIncludingNeighbor.begin());

	auxVecs.bucketKeysExpanded.erase(
			auxVecs.bucketKeysExpanded.begin() + numberInsideRange,
			auxVecs.bucketKeysExpanded.end());

	auxVecs.bucketValuesIncludingNeighbor.erase(
			auxVecs.bucketValuesIncludingNeighbor.begin() + numberInsideRange,
			auxVecs.bucketValuesIncludingNeighbor.end());




	thrust::counting_iterator<unsigned> search_begin(0);

	thrust::lower_bound(auxVecs.bucketKeysExpanded.begin(),
		auxVecs.bucketKeysExpanded.end(), search_begin,
		search_begin + domainParams.totalBucketCount,
		auxVecs.keyBegin.begin());

	thrust::upper_bound(auxVecs.bucketKeysExpanded.begin(),
		auxVecs.bucketKeysExpanded.end(),search_begin,
		search_begin + domainParams.totalBucketCount,
		auxVecs.keyEnd.begin());

	//platelets
	unsigned valuesPltCount = auxVecs.bucketPltValues.size();
	thrust::fill(auxVecs.bucketPltKeysExpanded.begin(),auxVecs.bucketPltKeysExpanded.end(),0);
	thrust::fill(auxVecs.bucketPltValuesIncludingNeighbor.begin(),auxVecs.bucketPltValuesIncludingNeighbor.end(),0);




	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> pltfirst(27);
	/**
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/
	thrust::constant_iterator<unsigned> pltlast = pltfirst + (auxVecs.endIndexBucketPltKeys); // this is NOT numerical addition!

	expand(pltfirst, pltlast,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketPltKeys.begin(),
				auxVecs.bucketPltValues.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketPltKeysExpanded.begin(),
				auxVecs.bucketPltValuesIncludingNeighbor.begin())));


	thrust::counting_iterator<unsigned> pltcountingBegin(0);

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketPltKeysExpanded.begin(),
				pltcountingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketPltKeysExpanded.begin(),
				pltcountingBegin)) + endIndexPltExpanded,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.bucketPltKeysExpanded.begin(),
				pltcountingBegin)),
		NeighborFunctor(
			domainParams.XBucketCount,
			domainParams.YBucketCount,
			domainParams.ZBucketCount));



	unsigned pltnumberOfOutOfRange = thrust::count_if(auxVecs.bucketPltKeysExpanded.begin(),
		auxVecs.bucketPltKeysExpanded.end(), is_greater_than(domainParams.totalBucketCount) );
	unsigned pltnumberInsideRange = endIndexPltExpanded - pltnumberOfOutOfRange;

	//unsigned endIndexPltSearch = endIndexPltExpanded - pltnumberOfOutOfRange;

	thrust::sort_by_key(auxVecs.bucketPltKeysExpanded.begin(),
		auxVecs.bucketPltKeysExpanded.begin() + endIndexPltExpanded,
		auxVecs.bucketPltValuesIncludingNeighbor.begin());

	auxVecs.bucketPltKeysExpanded.erase(
			auxVecs.bucketPltKeysExpanded.begin() + pltnumberInsideRange,
			auxVecs.bucketPltKeysExpanded.end());

	auxVecs.bucketPltValuesIncludingNeighbor.erase(
			auxVecs.bucketPltValuesIncludingNeighbor.begin() + pltnumberInsideRange,
			auxVecs.bucketPltValuesIncludingNeighbor.end());




	thrust::counting_iterator<unsigned> pltsearch_begin(0);

	thrust::lower_bound(auxVecs.bucketPltKeysExpanded.begin(),
		auxVecs.bucketPltKeysExpanded.end(), pltsearch_begin,
		pltsearch_begin + domainParams.totalBucketCount,
		auxVecs.keyPltBegin.begin());

	thrust::upper_bound(auxVecs.bucketPltKeysExpanded.begin(),
		auxVecs.bucketPltKeysExpanded.end(),pltsearch_begin,
		pltsearch_begin + domainParams.totalBucketCount,
		auxVecs.keyPltEnd.begin());

}


//takes nodes and places in buckets.
void buildBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {


	thrust::counting_iterator<unsigned> indexBucketBegin(0);
	// takes counting iterator and coordinates
	// return tuple of keys and values
	// transform the points to their bucket indices
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)) + generalParams.maxNodeCount,
		BucketIndexer(
			domainParams.minX, domainParams.maxX, domainParams.minY,
			domainParams.maxY, domainParams.minZ, domainParams.maxZ,
			domainParams.gridSpacing,
			thrust::raw_pointer_cast(auxVecs.bucketKeys.data()),
			thrust::raw_pointer_cast(auxVecs.bucketValues.data())));

//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.bucketValues.begin(),
		auxVecs.bucketValues.begin() + generalParams.maxNodeCount,
		auxVecs.bucketKeys.begin());
	unsigned numberOutOfRange = thrust::count(auxVecs.bucketKeys.begin(),
			auxVecs.bucketKeys.begin() + generalParams.maxNodeCount, ULONG_MAX);

	auxVecs.endIndexBucketKeys = generalParams.maxNodeCount - numberOutOfRange;

	//platelets
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				pltInfoVecs.pltLocX.begin(),
				pltInfoVecs.pltLocY.begin(),
				pltInfoVecs.pltLocZ.begin(),
				indexBucketBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				pltInfoVecs.pltLocX.begin(),
				pltInfoVecs.pltLocY.begin(),
				pltInfoVecs.pltLocZ.begin(),
				indexBucketBegin)) + generalParams.maxPltCount,
		BucketIndexer(
			domainParams.minX, domainParams.maxX, domainParams.minY,
			domainParams.maxY, domainParams.minZ, domainParams.maxZ,
			domainParams.gridSpacing,
			thrust::raw_pointer_cast(auxVecs.bucketPltKeys.data()),
			thrust::raw_pointer_cast(auxVecs.bucketPltValues.data())));

//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.bucketPltValues.begin(),
		auxVecs.bucketPltValues.begin() + generalParams.maxPltCount,
		auxVecs.bucketPltKeys.begin());
	unsigned numberPltOutOfRange = thrust::count(auxVecs.bucketPltKeys.begin(),
			auxVecs.bucketPltKeys.begin() + generalParams.maxPltCount, ULONG_MAX);

	auxVecs.endIndexBucketPltKeys = generalParams.maxPltCount - numberPltOutOfRange;


};
