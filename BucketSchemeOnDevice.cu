#include "hip/hip_runtime.h"

#include "BucketSchemeOnDevice.h"
#include "NodeSystemDevice.h"

//take domain and discretize into square buckets of size gridspace
void initDimensionBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	double minXTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocX.begin(), nodeInfoVecs.nodeLocX.end())));
	double maxXTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocX.begin(), nodeInfoVecs.nodeLocX.end())));
	double minYTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocY.begin(), nodeInfoVecs.nodeLocY.end())));
	double maxYTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocY.begin(), nodeInfoVecs.nodeLocY.end())));
	double minZTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocZ.begin(), nodeInfoVecs.nodeLocZ.end())));
	double maxZTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocZ.begin(), nodeInfoVecs.nodeLocZ.end())));

	//platelets
	domainParams.pltminX = (*(thrust::min_element(pltInfoVecs.pltLocX.begin(), pltInfoVecs.pltLocX.end())));
	domainParams.pltmaxX = (*(thrust::max_element(pltInfoVecs.pltLocX.begin(), pltInfoVecs.pltLocX.end())));
	domainParams.pltminY = (*(thrust::min_element(pltInfoVecs.pltLocY.begin(), pltInfoVecs.pltLocY.end())));
	domainParams.pltmaxY = (*(thrust::max_element(pltInfoVecs.pltLocY.begin(), pltInfoVecs.pltLocY.end())));
	domainParams.pltminZ = (*(thrust::min_element(pltInfoVecs.pltLocZ.begin(), pltInfoVecs.pltLocZ.end())));
	domainParams.pltmaxZ = (*(thrust::max_element(pltInfoVecs.pltLocZ.begin(), pltInfoVecs.pltLocZ.end())));

	double space = 0.0;
	domainParams.minX = min(minXTemp, domainParams.pltminX) - space;
	domainParams.maxX = max(maxXTemp, domainParams.pltmaxX) + space;
	domainParams.minY = min(minYTemp, domainParams.pltminY) - space;
	domainParams.maxY = max(maxYTemp, domainParams.pltmaxY) + space;
	domainParams.minZ = min(minZTemp, domainParams.pltminZ) - space;
	domainParams.maxZ = max(maxZTemp, domainParams.pltmaxZ) + space;

	domainParams.XBucketCount = (ceil(domainParams.maxX - domainParams.minX) / domainParams.gridSpacing) + 1;
	domainParams.YBucketCount = (ceil(domainParams.maxY - domainParams.minY) / domainParams.gridSpacing) + 1;
	domainParams.ZBucketCount = (ceil(domainParams.maxZ - domainParams.minZ) / domainParams.gridSpacing) + 1;

	if ( (domainParams.XBucketCount * domainParams.YBucketCount * domainParams.ZBucketCount) != domainParams.totalBucketCount	) {

		//double amount of buckets in case of resizing networks
		domainParams.totalBucketCount = domainParams.XBucketCount * domainParams.YBucketCount * domainParams.ZBucketCount;
		std::cout<<"grid: "<< domainParams.gridSpacing << std::endl;
		std::cout<<"total bucket count: "<< domainParams.totalBucketCount<<std::endl;

		auxVecs.keyBegin.resize(domainParams.totalBucketCount);
		auxVecs.keyEnd.resize(domainParams.totalBucketCount);
		//platelets
		auxVecs.keyPltBegin.resize(domainParams.totalBucketCount);
		auxVecs.keyPltEnd.resize(domainParams.totalBucketCount);

	}

	thrust::fill(auxVecs.keyBegin.begin(),auxVecs.keyBegin.end(),0);
	thrust::fill(auxVecs.keyEnd.begin(),auxVecs.keyEnd.end(),0);
	//platelets
	thrust::fill(auxVecs.keyPltBegin.begin(),auxVecs.keyPltBegin.end(),0);
	thrust::fill(auxVecs.keyPltEnd.begin(),auxVecs.keyPltEnd.end(),0);

};

//convert buckets into neighboring scheme
void extendBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs) {

	//memory is already allocated.
	unsigned endIndexExpanded = (auxVecs.endIndexBucketKeys) * 27;
	//platelets
	unsigned endIndexPltExpanded = (auxVecs.endIndexBucketPltKeys) * 27;

	//test for removing copies.
	unsigned valuesCount = auxVecs.id_value.size();
	thrust::fill(auxVecs.id_bucket_expanded.begin(),auxVecs.id_bucket_expanded.end(),0);
	thrust::fill(auxVecs.id_value_expanded.begin(),auxVecs.id_value_expanded.end(),0);

	thrust::fill(auxVecs.idPlt_bucket_expanded.begin(),auxVecs.idPlt_bucket_expanded.end(),0);
	thrust::fill(auxVecs.idPlt_value_expanded.begin(),auxVecs.idPlt_value_expanded.end(),0);




	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> first(27);
	/**
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/
	thrust::constant_iterator<unsigned> last = first + (auxVecs.endIndexBucketKeys); // this is NOT numerical addition!

	expand(first, last,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket.begin(),
				auxVecs.id_value.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded.begin(),
				auxVecs.id_value_expanded.begin())));


	thrust::counting_iterator<unsigned> countingBegin(0);

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded.begin(),
				countingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded.begin(),
				countingBegin)) + endIndexExpanded,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded.begin(),
				countingBegin)),
		NeighborFunctor(
			domainParams.XBucketCount,
			domainParams.YBucketCount,
			domainParams.ZBucketCount));



	unsigned numberOfOutOfRange = thrust::count_if(auxVecs.id_bucket_expanded.begin(),
		auxVecs.id_bucket_expanded.end(), is_greater_than(domainParams.totalBucketCount) );
	unsigned numberInsideRange = endIndexExpanded - numberOfOutOfRange;

	//unsigned endIndexSearch = endIndexExpanded - numberOfOutOfRange;

	thrust::stable_sort_by_key(auxVecs.id_bucket_expanded.begin(),
		auxVecs.id_bucket_expanded.begin() + endIndexExpanded,
		auxVecs.id_value_expanded.begin());

	numberInsideRange =
		thrust::get<0>(thrust::unique_by_key(auxVecs.id_value_expanded.begin(),
			auxVecs.id_value_expanded.begin() + endIndexExpanded,
			auxVecs.id_bucket_expanded.begin())) - auxVecs.id_value_expanded.begin();

	auxVecs.id_bucket_expanded.erase(
			auxVecs.id_bucket_expanded.begin() + numberInsideRange,
			auxVecs.id_bucket_expanded.end());

	auxVecs.id_value_expanded.erase(
			auxVecs.id_value_expanded.begin() + numberInsideRange,
			auxVecs.id_value_expanded.end());



	thrust::counting_iterator<unsigned> search_begin(0);

	thrust::lower_bound(auxVecs.id_bucket_expanded.begin(),
		auxVecs.id_bucket_expanded.end(), search_begin,
		search_begin + domainParams.totalBucketCount,
		auxVecs.keyBegin.begin());

	thrust::upper_bound(auxVecs.id_bucket_expanded.begin(),
		auxVecs.id_bucket_expanded.end(),search_begin,
		search_begin + domainParams.totalBucketCount,
		auxVecs.keyEnd.begin());

	//platelets
	/*unsigned valuesPltCount = auxVecs.idPlt_value.size();
	thrust::fill(auxVecs.idPlt_bucket_expanded.begin(),auxVecs.idPlt_bucket_expanded.end(),0);
	thrust::fill(auxVecs.idPlt_value_expanded.begin(),auxVecs.idPlt_value_expanded.end(),0);
*/



	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> pltfirst(27);
	/**
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/


	thrust::constant_iterator<unsigned> pltlast = pltfirst + (auxVecs.endIndexBucketPltKeys); // this is NOT numerical addition!

	expand(pltfirst, pltlast,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket.begin(),
				auxVecs.idPlt_value.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket_expanded.begin(),
				auxVecs.idPlt_value_expanded.begin())));


	thrust::counting_iterator<unsigned> pltcountingBegin(0);

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket_expanded.begin(),
				pltcountingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket_expanded.begin(),
				pltcountingBegin)) + endIndexPltExpanded,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket_expanded.begin(),
				pltcountingBegin)),
		NeighborFunctor(
			domainParams.XBucketCount,
			domainParams.YBucketCount,
			domainParams.ZBucketCount));



	unsigned pltnumberOfOutOfRange = thrust::count_if(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.end(), is_greater_than(domainParams.totalBucketCount) );
	unsigned pltnumberInsideRange = endIndexPltExpanded - pltnumberOfOutOfRange;

	//unsigned endIndexPltSearch = endIndexPltExpanded - pltnumberOfOutOfRange;

	thrust::sort_by_key(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.begin() + endIndexPltExpanded,
		auxVecs.idPlt_value_expanded.begin());

	pltnumberInsideRange =
		thrust::get<0>(thrust::unique_by_key(auxVecs.idPlt_value_expanded.begin(),
			auxVecs.idPlt_value_expanded.begin() + endIndexExpanded,
			auxVecs.idPlt_bucket_expanded.begin())) - auxVecs.idPlt_value_expanded.begin();

	auxVecs.idPlt_bucket_expanded.erase(
			auxVecs.idPlt_bucket_expanded.begin() + pltnumberInsideRange,
			auxVecs.idPlt_bucket_expanded.end());

	auxVecs.idPlt_value_expanded.erase(
			auxVecs.idPlt_value_expanded.begin() + pltnumberInsideRange,
			auxVecs.idPlt_value_expanded.end());




	thrust::counting_iterator<unsigned> pltsearch_begin(0);

	thrust::lower_bound(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.end(), pltsearch_begin,
		pltsearch_begin + domainParams.totalBucketCount,
		auxVecs.keyPltBegin.begin());

	thrust::upper_bound(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.end(),pltsearch_begin,
		pltsearch_begin + domainParams.totalBucketCount,
		auxVecs.keyPltEnd.begin());

}


//takes nodes and places in buckets.
void buildBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {


	thrust::counting_iterator<unsigned> indexBucketBegin(0);
	// takes counting iterator and coordinates
	// return tuple of keys and values
	// transform the points to their bucket indices

	//std::cout<<"bucket nodes"<<std::endl;
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)) + generalParams.maxNodeCount,
		BucketIndexer(
			domainParams.minX, domainParams.maxX, domainParams.minY,
			domainParams.maxY, domainParams.minZ, domainParams.maxZ,
			domainParams.gridSpacing,
			thrust::raw_pointer_cast(auxVecs.id_bucket.data()),
			thrust::raw_pointer_cast(auxVecs.id_value.data())));

//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.id_value.begin(),
		auxVecs.id_value.begin() + generalParams.maxNodeCount,
		auxVecs.id_bucket.begin());
unsigned numberOutOfRange = thrust::count(auxVecs.id_bucket.begin(),
			auxVecs.id_bucket.begin() + generalParams.maxNodeCount, ULONG_MAX);

	auxVecs.endIndexBucketKeys = generalParams.maxNodeCount - numberOutOfRange;

	//platelets
	//std::cout<<"bucket platelet"<<std::endl;
	thrust::counting_iterator<unsigned> indexBucketBegin1(0);
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				pltInfoVecs.pltLocX.begin(),
				pltInfoVecs.pltLocY.begin(),
				pltInfoVecs.pltLocZ.begin(),
				indexBucketBegin1)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				pltInfoVecs.pltLocX.begin(),
				pltInfoVecs.pltLocY.begin(),
				pltInfoVecs.pltLocZ.begin(),
				indexBucketBegin1)) + generalParams.maxPltCount,
		BucketIndexer(
			domainParams.minX, domainParams.maxX, domainParams.minY,
			domainParams.maxY, domainParams.minZ, domainParams.maxZ,
			domainParams.gridSpacing,
			thrust::raw_pointer_cast(auxVecs.idPlt_bucket.data()),
			thrust::raw_pointer_cast(auxVecs.idPlt_value.data())));


	//std::cout<<"end bucket platelet"<<std::endl;
//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.idPlt_value.begin(),
		auxVecs.idPlt_value.begin() + generalParams.maxPltCount,
		auxVecs.idPlt_bucket.begin());

unsigned numberPltOutOfRange = thrust::count(auxVecs.idPlt_bucket.begin(),
			auxVecs.idPlt_bucket.begin() + generalParams.maxPltCount, ULONG_MAX);

	auxVecs.endIndexBucketPltKeys = generalParams.maxPltCount - numberPltOutOfRange;


};
