#include "hip/hip_runtime.h"

#include "BucketSchemeOnDevice.h"
#include "NodeSystemDevice.h"

//take domain and discretize into square buckets of size gridspace
void initDimensionBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	double minXTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocX.begin(), nodeInfoVecs.nodeLocX.end())));
	double maxXTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocX.begin(), nodeInfoVecs.nodeLocX.end())));
	double minYTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocY.begin(), nodeInfoVecs.nodeLocY.end())));
	double maxYTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocY.begin(), nodeInfoVecs.nodeLocY.end())));
	double minZTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocZ.begin(), nodeInfoVecs.nodeLocZ.end())));
	double maxZTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocZ.begin(), nodeInfoVecs.nodeLocZ.end())));

	//platelets
	domainParams.pltminX = (*(thrust::min_element(pltInfoVecs.pltLocX.begin(), pltInfoVecs.pltLocX.end())));
	domainParams.pltmaxX = (*(thrust::max_element(pltInfoVecs.pltLocX.begin(), pltInfoVecs.pltLocX.end())));
	domainParams.pltminY = (*(thrust::min_element(pltInfoVecs.pltLocY.begin(), pltInfoVecs.pltLocY.end())));
	domainParams.pltmaxY = (*(thrust::max_element(pltInfoVecs.pltLocY.begin(), pltInfoVecs.pltLocY.end())));
	domainParams.pltminZ = (*(thrust::min_element(pltInfoVecs.pltLocZ.begin(), pltInfoVecs.pltLocZ.end())));
	domainParams.pltmaxZ = (*(thrust::max_element(pltInfoVecs.pltLocZ.begin(), pltInfoVecs.pltLocZ.end())));

	double space = 0.0;
	domainParams.minX = min(minXTemp, domainParams.pltminX) - space;
	domainParams.maxX = max(maxXTemp, domainParams.pltmaxX) + space;
	domainParams.minY = min(minYTemp, domainParams.pltminY) - space;
	domainParams.maxY = max(maxYTemp, domainParams.pltmaxY) + space;
	domainParams.minZ = min(minZTemp, domainParams.pltminZ) - space;
	domainParams.maxZ = max(maxZTemp, domainParams.pltmaxZ) + space;

	domainParams.XBucketCount = ceil((domainParams.maxX - domainParams.minX) / domainParams.gridSpacing) + 1;
	domainParams.YBucketCount = ceil((domainParams.maxY - domainParams.minY) / domainParams.gridSpacing) + 1;
	domainParams.ZBucketCount = ceil((domainParams.maxZ - domainParams.minZ) / domainParams.gridSpacing) + 1;

	if ( (domainParams.XBucketCount * domainParams.YBucketCount * domainParams.ZBucketCount) != domainParams.totalBucketCount	) {

		//double amount of buckets in case of resizing networks
		domainParams.totalBucketCount = domainParams.XBucketCount * domainParams.YBucketCount * domainParams.ZBucketCount;
		std::cout<<"grid: "<< domainParams.gridSpacing << std::endl;
		std::cout<<"total bucket count: "<< domainParams.totalBucketCount<<std::endl;

		auxVecs.keyBegin.resize(domainParams.totalBucketCount);
		auxVecs.keyEnd.resize(domainParams.totalBucketCount);
		//platelets
		auxVecs.keyPltBegin.resize(domainParams.totalBucketCount); 
		auxVecs.keyPltEnd.resize(domainParams.totalBucketCount); 
 
	}

	thrust::fill(auxVecs.keyBegin.begin(),auxVecs.keyBegin.end(),0);
	thrust::fill(auxVecs.keyEnd.begin(),auxVecs.keyEnd.end(),0);
	//platelets
	thrust::fill(auxVecs.keyPltBegin.begin(),auxVecs.keyPltBegin.end(),0);
	thrust::fill(auxVecs.keyPltEnd.begin(),auxVecs.keyPltEnd.end(),0);

};

//convert buckets into neighboring scheme
void extendBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs) {

	//memory is already allocated.
	unsigned endIndexExpanded = (auxVecs.endIndexBucketKeys) * 27;
	//platelets
	unsigned endIndexPltExpanded = (auxVecs.endIndexBucketPltKeys) * 27;

	//test for removing copies.
	unsigned valuesCount = auxVecs.id_value.size();
	thrust::fill(auxVecs.id_bucket_expanded.begin(),auxVecs.id_bucket_expanded.end(),0);
	thrust::fill(auxVecs.id_value_expanded.begin(),auxVecs.id_value_expanded.end(),0);

	thrust::fill(auxVecs.idPlt_bucket_expanded.begin(),auxVecs.idPlt_bucket_expanded.end(),0);
	thrust::fill(auxVecs.idPlt_value_expanded.begin(),auxVecs.idPlt_value_expanded.end(),0);




	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> first(27);
	/*
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/
	thrust::constant_iterator<unsigned> last = first + (auxVecs.endIndexBucketKeys); // this is NOT numerical addition!

	expand(first, last,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket.begin(),
				auxVecs.id_value.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded.begin(),
				auxVecs.id_value_expanded.begin())));

	thrust::counting_iterator<unsigned> countingBegin(0);
 
	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded.begin(),
				countingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded.begin(),
				countingBegin)) + endIndexExpanded,
		
		auxVecs.id_bucket_expanded.begin(),
		NeighborFunctor(
			domainParams.XBucketCount,
			domainParams.YBucketCount,
			domainParams.ZBucketCount)); 

/*	unsigned choice = 479;

	for (unsigned i = choice*27; i < choice*27+27; i++)
		std::cout<<auxVecs.id_bucket_expanded[i]<< " "<< auxVecs.id_value_expanded[i] <<std::endl;
*/
	unsigned numberOfOutOfRange = thrust::count_if(auxVecs.id_bucket_expanded.begin(),
		auxVecs.id_bucket_expanded.end(), is_greater_than(domainParams.totalBucketCount) );
	unsigned numberInsideRange = endIndexExpanded - numberOfOutOfRange;

	__attribute__ ((unused)) unsigned endIndexSearch = endIndexExpanded - numberOfOutOfRange;

	thrust::stable_sort_by_key(auxVecs.id_bucket_expanded.begin(),
		auxVecs.id_bucket_expanded.begin() + endIndexExpanded,
		auxVecs.id_value_expanded.begin());

	numberInsideRange =
		thrust::get<0>(thrust::unique_by_key(auxVecs.id_value_expanded.begin(),
			auxVecs.id_value_expanded.begin() + endIndexExpanded,
			auxVecs.id_bucket_expanded.begin())) - auxVecs.id_value_expanded.begin();

	auxVecs.id_bucket_expanded.erase(
			auxVecs.id_bucket_expanded.begin() + numberInsideRange, 
			auxVecs.id_bucket_expanded.end());

	auxVecs.id_value_expanded.erase(
			auxVecs.id_value_expanded.begin() + numberInsideRange,
			auxVecs.id_value_expanded.end());  



	thrust::counting_iterator<unsigned> search_begin(0);

	thrust::lower_bound(auxVecs.id_bucket_expanded.begin(),
		auxVecs.id_bucket_expanded.end(), search_begin,
		search_begin + domainParams.totalBucketCount,
		auxVecs.keyBegin.begin());

	thrust::upper_bound(auxVecs.id_bucket_expanded.begin(),
		auxVecs.id_bucket_expanded.end(),search_begin,
		search_begin + domainParams.totalBucketCount,
		auxVecs.keyEnd.begin());

/*	unsigned bucket = auxVecs.id_bucket[choice];
	unsigned begin = auxVecs.keyBegin[bucket];
	unsigned end = auxVecs.keyEnd[bucket];
	
	std::cout<<"from bucket scheme:"<<std::endl;
	for (unsigned i = begin; i < end; i++){
		
		unsigned nbr = auxVecs.id_value_expanded[i];
		unsigned buck = auxVecs.id_bucket[nbr];
		double x_dist=nodeInfoVecs.nodeLocX[nbr] - nodeInfoVecs.nodeLocX[choice];
		double y_dist=nodeInfoVecs.nodeLocY[nbr] - nodeInfoVecs.nodeLocY[choice];
		double z_dist=nodeInfoVecs.nodeLocZ[nbr] - nodeInfoVecs.nodeLocZ[choice];
		double dist = std::sqrt(std::pow(x_dist,2.0)+std::pow(y_dist,2.0)+std::pow(z_dist,2.0));
		if (dist < 1.0){
			std::cout<<"dist: "<< dist<< " between: "<< choice << " and nbr: "<< nbr<<std::endl; 
			std::cout<<"nbr: "<< nbr<< " is in bucket: "<< buck <<std::endl;
		}
	}
	std::cout<<"from all nodes:"<<std::endl;
	for (unsigned i = 0; i < 3988; i++){
		unsigned nbr = i;//auxVecs.id_value_expanded[i];
		unsigned buck = auxVecs.id_bucket[nbr];
		double x_dist=nodeInfoVecs.nodeLocX[nbr] - nodeInfoVecs.nodeLocX[choice];
		double y_dist=nodeInfoVecs.nodeLocY[nbr] - nodeInfoVecs.nodeLocY[choice];
		double z_dist=nodeInfoVecs.nodeLocZ[nbr] - nodeInfoVecs.nodeLocZ[choice];
		double dist = std::sqrt(std::pow(x_dist,2.0)+std::pow(y_dist,2.0)+std::pow(z_dist,2.0));
		if (dist < 1.0){
			std::cout<<"dist: "<< dist<< " between: "<< choice << " and nbr: "<< nbr<<std::endl; 
			std::cout<<"nbr: "<< nbr<< " is in bucket: "<< buck <<std::endl;
		} 
	} */
  
	//platelets 
	/*unsigned valuesPltCount = auxVecs.idPlt_value.size();
	thrust::fill(auxVecs.idPlt_bucket_expanded.begin(),auxVecs.idPlt_bucket_expanded.end(),0);
	thrust::fill(auxVecs.idPlt_value_expanded.begin(),auxVecs.idPlt_value_expanded.end(),0);
*/



	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> pltfirst(27);
	/**
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/


	thrust::constant_iterator<unsigned> pltlast = pltfirst + (auxVecs.endIndexBucketPltKeys); // this is NOT numerical addition!

	expand(pltfirst, pltlast,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket.begin(),
				auxVecs.idPlt_value.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket_expanded.begin(),
				auxVecs.idPlt_value_expanded.begin())));


	thrust::counting_iterator<unsigned> pltcountingBegin(0);

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket_expanded.begin(),
				pltcountingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket_expanded.begin(),
				pltcountingBegin)) + endIndexPltExpanded,
			auxVecs.idPlt_bucket_expanded.begin(),
		NeighborFunctor(
			domainParams.XBucketCount,
			domainParams.YBucketCount,
			domainParams.ZBucketCount));



	unsigned pltnumberOfOutOfRange = thrust::count_if(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.end(), is_greater_than(domainParams.totalBucketCount) );
	unsigned pltnumberInsideRange = endIndexPltExpanded - pltnumberOfOutOfRange;

	//unsigned endIndexPltSearch = endIndexPltExpanded - pltnumberOfOutOfRange;

	thrust::sort_by_key(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.begin() + endIndexPltExpanded,
		auxVecs.idPlt_value_expanded.begin());

	pltnumberInsideRange =
		thrust::get<0>(thrust::unique_by_key(auxVecs.idPlt_value_expanded.begin(),
			auxVecs.idPlt_value_expanded.begin() + endIndexExpanded,
			auxVecs.idPlt_bucket_expanded.begin())) - auxVecs.idPlt_value_expanded.begin();

	auxVecs.idPlt_bucket_expanded.erase(
			auxVecs.idPlt_bucket_expanded.begin() + pltnumberInsideRange,
			auxVecs.idPlt_bucket_expanded.end());

	auxVecs.idPlt_value_expanded.erase(
			auxVecs.idPlt_value_expanded.begin() + pltnumberInsideRange,
			auxVecs.idPlt_value_expanded.end());




	thrust::counting_iterator<unsigned> pltsearch_begin(0);

	thrust::lower_bound(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.end(), pltsearch_begin,
		pltsearch_begin + domainParams.totalBucketCount,
		auxVecs.keyPltBegin.begin());

	thrust::upper_bound(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.end(),pltsearch_begin,
		pltsearch_begin + domainParams.totalBucketCount,
		auxVecs.keyPltEnd.begin());

}


//takes nodes and places in buckets.
void buildBucketScheme(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {


	thrust::counting_iterator<unsigned> indexBucketBegin(0);
	// takes counting iterator and coordinates
	// return tuple of keys and values
	// transform the points to their bucket indices

	//std::cout<<"bucket nodes"<<std::endl;

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)) + generalParams.maxNodeCount,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket.begin(),
				auxVecs.id_value.begin())),
		BucketIndexer(
			domainParams.minX, domainParams.maxX, domainParams.minY,
			domainParams.maxY, domainParams.minZ, domainParams.maxZ,
			domainParams.XBucketCount,domainParams.YBucketCount,domainParams.ZBucketCount,
			domainParams.gridSpacing));

//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.id_value.begin(),
		auxVecs.id_value.begin() + generalParams.maxNodeCount,
		auxVecs.id_bucket.begin());
unsigned numberOutOfRange = thrust::count(auxVecs.id_bucket.begin(),
			auxVecs.id_bucket.begin() + generalParams.maxNodeCount, ULONG_MAX);

	auxVecs.endIndexBucketKeys = generalParams.maxNodeCount - numberOutOfRange;

	//platelets
	//std::cout<<"bucket platelet"<<std::endl;
	thrust::counting_iterator<unsigned> indexBucketBegin1(0);
	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				pltInfoVecs.pltLocX.begin(),
				pltInfoVecs.pltLocY.begin(),
				pltInfoVecs.pltLocZ.begin(),
				indexBucketBegin1)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				pltInfoVecs.pltLocX.begin(),
				pltInfoVecs.pltLocY.begin(),
				pltInfoVecs.pltLocZ.begin(),
				indexBucketBegin1)) + generalParams.maxPltCount,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket.begin(),
				auxVecs.idPlt_value.begin())),
		BucketIndexer(
			domainParams.minX, domainParams.maxX, domainParams.minY,
			domainParams.maxY, domainParams.minZ, domainParams.maxZ,
			domainParams.XBucketCount,domainParams.YBucketCount,domainParams.ZBucketCount,
			domainParams.gridSpacing));


	//std::cout<<"end bucket platelet"<<std::endl;
//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.idPlt_value.begin(),
		auxVecs.idPlt_value.begin() + generalParams.maxPltCount,
		auxVecs.idPlt_bucket.begin());

unsigned numberPltOutOfRange = thrust::count(auxVecs.idPlt_bucket.begin(),
			auxVecs.idPlt_bucket.begin() + generalParams.maxPltCount, ULONG_MAX);

	auxVecs.endIndexBucketPltKeys = generalParams.maxPltCount - numberPltOutOfRange;


};
