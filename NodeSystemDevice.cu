#include "hip/hip_runtime.h"
#include "ForceDiagramStorage.h"
#include "LinkNodesOnDevice.h"
#include "WLCSolveOnDevice.h"
#include "TorsionSolveOnDevice.h"
#include "PltForceDevice.h"
#include "PltForceFieldDevice.h"
#include "PltonPltForceFieldDevice.h"
#include "PltTndrlDevice.h"
#include "PltonPltTndrlDevice.h"
#include "PltVlmPushDevice.h"
#include "AdvancePositionOnDevice.h"
#include "BucketSchemeOnDevice.h"
#include "NodeSystemDevice.h"



void NodeSystemDevice::setBucketScheme() {

	initDimensionBucketScheme(
		nodeInfoVecs,
		pltInfoVecs,
		domainParams,
		auxVecs,
		generalParams);

	buildBucketScheme(nodeInfoVecs, pltInfoVecs, domainParams,
		auxVecs, generalParams);

	extendBucketScheme(nodeInfoVecs, pltInfoVecs, domainParams, auxVecs);
};

void NodeSystemDevice::solveForcesOnDevice() {

	//RESET FORCE TO ZERO AT BEGINNING/////////////////////////////////////////////////
	thrust::fill(nodeInfoVecs.nodeForceX.begin(),nodeInfoVecs.nodeForceX.end(),0);
	thrust::fill(nodeInfoVecs.nodeForceY.begin(),nodeInfoVecs.nodeForceY.end(),0);
	thrust::fill(nodeInfoVecs.nodeForceZ.begin(),nodeInfoVecs.nodeForceZ.end(),0);

	if (generalParams.linking == true) {
			LinkNodesOnDevice(
					nodeInfoVecs,
					wlcInfoVecs,
					auxVecs,
					generalParams);
	}
	TorsionSolveOnDevice(nodeInfoVecs, torsionInfoVecs, generalParams);

	//std::cout<<"prewlc"<<std::endl;
	WLCSolveOnDevice(nodeInfoVecs, wlcInfoVecs, generalParams);

	//platetelet-node forces
	//RESETS PLATELET FORCES
	if (generalParams.pltfrcfld == true) {// note: this force-field includes both pulling and pushing
		PltForceFieldOnDevice(//platelet on node force field
			nodeInfoVecs,
			wlcInfoVecs,
			generalParams,
			pltInfoVecs,
			auxVecs);
		if (generalParams.pltonplt == true) {
			PltInteractionPltOnDevice(//platelet on platelet interaction through force field
				generalParams,
				pltInfoVecs,
				auxVecs);
		}

	}
	else if (generalParams.plttndrl == true) { //note for now force-field type has priority over tndrl-type
		//initialize Trnl-Node Id list
	  if (generalParams.currentTime==0.0){
	    thrust::fill(pltInfoVecs.tndrlNodeId.begin(),pltInfoVecs.tndrlNodeId.end(), generalParams.maxIdCount);
			thrust::fill(pltInfoVecs.tndrlNodeType.begin(),pltInfoVecs.tndrlNodeType.end(), 0);
	    }

		// Tndrl-node pulling
		PltTndrlOnDevice(
		  nodeInfoVecs,
		  wlcInfoVecs,
		  generalParams,
		  pltInfoVecs,
		  auxVecs);

		//Tndrl-Plt pulling
		if (generalParams.pltonplt == true) {
			PltonPltTndrlOnDevice(//platelet on platelet interaction through tndrl
				generalParams,
				pltInfoVecs,
				auxVecs);
		}

		PltVlmPushOnDevice(//push for volume exclusion
			nodeInfoVecs,
			wlcInfoVecs,
			generalParams,
			pltInfoVecs,
			auxVecs);

	}




};


void NodeSystemDevice::solveSystemDevice() {


	//set initial epsilon
	generalParams.epsilon = (1.0) *
		sqrt(6.0*generalParams.kB * generalParams.temperature * generalParams.dtTemp / generalParams.viscousDamp);

	while (generalParams.runSim == true) {
		generalParams.iterationCounter++;
		generalParams.currentTime += generalParams.dtTemp;



		AdvancePositionOnDevice(
			nodeInfoVecs,
			pltInfoVecs,
		 	generalParams);

		setBucketScheme();

		solveForcesOnDevice(); //resets and solves forces for next time step


		if (generalParams.iterationCounter % 50 == 0) {
			storage->print_VTK_File();
			//store sum of all forces on each node. Used in stress calculations
			//store before upadting storage class.
			thrust::transform(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.nodeForceX.begin(),
						nodeInfoVecs.nodeForceY.begin(),
						nodeInfoVecs.nodeForceZ.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.nodeForceX.begin(),
						nodeInfoVecs.nodeForceY.begin(),
						nodeInfoVecs.nodeForceZ.begin())) + generalParams.maxNodeCount,
				nodeInfoVecs.sumForcesOnNode.begin(),//save vector
				NormFunctor());
				//platelets
				thrust::transform(
					thrust::make_zip_iterator(
						thrust::make_tuple(
							pltInfoVecs.pltForceX.begin(),
							pltInfoVecs.pltForceY.begin(),
							pltInfoVecs.pltForceZ.begin())),
					thrust::make_zip_iterator(
						thrust::make_tuple(
							pltInfoVecs.pltForceX.begin(),
							pltInfoVecs.pltForceY.begin(),
							pltInfoVecs.pltForceZ.begin())) + generalParams.maxPltCount,
					pltInfoVecs.sumForcesOnPlt.begin(),//save vector
					NormFunctor());


			generalParams.epsilon = (1.0) *
				sqrt(6.0 * generalParams.kB * generalParams.temperature * generalParams.dtTemp / generalParams.viscousDamp);
		}

	}

};

NodeSystemDevice::NodeSystemDevice()  {};

void NodeSystemDevice::assignForceDiagramStorage(std::shared_ptr<ForceDiagramStorage> _storage) {
	storage = _storage;
}

//__host__ __device__
void NodeSystemDevice::initializeSystem(
	thrust::host_vector<bool>& hostIsNodeFixed,
	thrust::host_vector<double>& hostPosX,
	thrust::host_vector<double>& hostPosY,
	thrust::host_vector<double>& hostPosZ,
	thrust::host_vector<unsigned>& hostWLCEdgeLeft,
	thrust::host_vector<unsigned>& hostWLCEdgeRight,
	thrust::host_vector<double>& hostWLCLenZero,

	thrust::host_vector<unsigned>& hostWLCSubEdgeLeft,
	thrust::host_vector<unsigned>& hostWLCSubEdgeRight,
	thrust::host_vector<double>& hostWLCSubLenZero,
	thrust::host_vector<unsigned>& hostTorsionIndexLeft,
	thrust::host_vector<unsigned>& hostTorsionIndexCenter,
	thrust::host_vector<unsigned>& hostTorsionIndexRight,
	thrust::host_vector<double>& hostTorsionAngleZero,
	//platelets
	thrust::host_vector<bool>& hostIsPltFixed,
	thrust::host_vector<double>& hostPltPosX,
	thrust::host_vector<double>& hostPltPosY,
	thrust::host_vector<double>& hostPltPosZ) {

	std::cout<< "total Edge Count: "<< generalParams.originEdgeCount << std::endl;
	std::cout << "max num nodes: " << generalParams.maxNodeCount << std::endl;
	//platelets

	std::cout << "max num platelets in device: " << generalParams.maxPltCount << std::endl;



	setPltVecs(
		hostIsPltFixed,
		hostPltPosX,
		hostPltPosY,
		hostPltPosZ);

	setNodeVecs(//calls initDimensionBucketScheme
		hostIsNodeFixed,
		hostPosX,
		hostPosY,
		hostPosZ);


	setTorsionVecs(
		hostTorsionIndexLeft,
		hostTorsionIndexCenter,
		hostTorsionIndexRight,
		hostTorsionAngleZero);

	setWLCVecs(
		hostWLCEdgeLeft,
		hostWLCEdgeRight,
		hostWLCLenZero );

};

void NodeSystemDevice::setNodeVecs(
	thrust::host_vector<bool>& hostIsNodeFixed,
	thrust::host_vector<double>& hostPosX,
	thrust::host_vector<double>& hostPosY,
	thrust::host_vector<double>& hostPosZ) {


	nodeInfoVecs.sumForcesOnNode.resize(generalParams.maxNodeCount);

	nodeInfoVecs.nodeVelocity.resize(generalParams.maxNodeCount);

	nodeInfoVecs.nodeLocX.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeLocY.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeLocZ.resize(generalParams.maxNodeCount);

	nodeInfoVecs.nodeForceX.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeForceY.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeForceZ.resize(generalParams.maxNodeCount);

	nodeInfoVecs.discretizedEdgeStrain.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);
	nodeInfoVecs.discretizedEdgeAlignment.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);

	//sized larger for input later
	nodeInfoVecs.deviceEdgeLeft.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);
	nodeInfoVecs.deviceEdgeRight.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);


	thrust::fill(nodeInfoVecs.discretizedEdgeStrain.begin(), nodeInfoVecs.discretizedEdgeStrain.end(),0.0);
	thrust::fill(nodeInfoVecs.deviceEdgeRight.begin(), nodeInfoVecs.deviceEdgeRight.end(), 0);	//fill force and velocity with zeros for computation.
	thrust::fill(nodeInfoVecs.deviceEdgeLeft.begin(), nodeInfoVecs.deviceEdgeLeft.end(), 0);	//fill force and velocity with zeros for computation.

	thrust::fill(nodeInfoVecs.sumForcesOnNode.begin(), nodeInfoVecs.sumForcesOnNode.end(), 0);


	thrust::copy(hostPosX.begin(), hostPosX.end(), nodeInfoVecs.nodeLocX.begin());
	thrust::copy(hostPosY.begin(), hostPosY.end(), nodeInfoVecs.nodeLocY.begin());
	thrust::copy(hostPosZ.begin(), hostPosZ.end(), nodeInfoVecs.nodeLocZ.begin());


	//copy fixed positions
	nodeInfoVecs.isNodeFixed.resize(generalParams.maxNodeCount);
	thrust::copy(hostIsNodeFixed.begin(), hostIsNodeFixed.end(), nodeInfoVecs.isNodeFixed.begin());

	nodeInfoVecs.linksThreadMade.resize(generalParams.maxNodeCount);
	nodeInfoVecs.delinksThreadMade.resize(generalParams.maxNodeCount);
	nodeInfoVecs.idMadeTempLeft.resize(generalParams.maxNodeCount * generalParams.maxLinksPerIteration);
	nodeInfoVecs.idMadeTempRight.resize(generalParams.maxNodeCount * generalParams.maxLinksPerIteration);

	//at this point all nodes are filled, so we can generate domainParams
	initDimensionBucketScheme(
		nodeInfoVecs,
		pltInfoVecs,
		domainParams,
		auxVecs,
		generalParams);


	domainParams.originMinX = domainParams.minX;
	domainParams.originMaxX = domainParams.maxX;
	domainParams.originMinY = domainParams.minY;
	domainParams.originMaxY = domainParams.maxY;
	domainParams.originMinZ = domainParams.minZ;
	domainParams.originMaxZ = domainParams.maxZ;

	std::cout<< "node count : " <<nodeInfoVecs.nodeLocY.size()<< std::endl;


	auxVecs.id_bucket.resize(generalParams.maxNodeCount);
	auxVecs.id_value.resize(generalParams.maxNodeCount);
	auxVecs.id_bucket_expanded.resize(27 * (generalParams.maxNodeCount));
	auxVecs.id_value_expanded.resize(27 *( generalParams.maxNodeCount ));

};

//platelet
void NodeSystemDevice::setPltVecs(
	thrust::host_vector<bool>& hostIsPltFixed,
	thrust::host_vector<double>& hostPltPosX,
	thrust::host_vector<double>& hostPltPosY,
	thrust::host_vector<double>& hostPltPosZ) {


	pltInfoVecs.sumForcesOnPlt.resize(generalParams.maxPltCount);

	pltInfoVecs.pltVelocity.resize(generalParams.maxPltCount);

	pltInfoVecs.pltLocX.resize(generalParams.maxPltCount);
	pltInfoVecs.pltLocY.resize(generalParams.maxPltCount);
	pltInfoVecs.pltLocZ.resize(generalParams.maxPltCount);

	pltInfoVecs.pltForceX.resize(generalParams.maxPltCount);
	pltInfoVecs.pltForceY.resize(generalParams.maxPltCount);
	pltInfoVecs.pltForceZ.resize(generalParams.maxPltCount);

	pltInfoVecs.pltImagingConnection.resize(generalParams.maxPltCount * generalParams.pltMaxConn);
	pltInfoVecs.nodeImagingConnection.resize(generalParams.maxPltCount * generalParams.pltMaxConn);

	pltInfoVecs.nodeUnreducedId.resize(generalParams.maxPltCount * generalParams.pltMaxConn);
	pltInfoVecs.nodeUnreducedForceX.resize(generalParams.maxPltCount * generalParams.pltMaxConn);
	pltInfoVecs.nodeUnreducedForceY.resize(generalParams.maxPltCount * generalParams.pltMaxConn);
	pltInfoVecs.nodeUnreducedForceZ.resize(generalParams.maxPltCount * generalParams.pltMaxConn);

	pltInfoVecs.nodeReducedId.resize(generalParams.maxPltCount * generalParams.pltMaxConn);
	pltInfoVecs.nodeReducedForceX.resize(generalParams.maxPltCount * generalParams.pltMaxConn);
	pltInfoVecs.nodeReducedForceY.resize(generalParams.maxPltCount * generalParams.pltMaxConn);
	pltInfoVecs.nodeReducedForceZ.resize(generalParams.maxPltCount * generalParams.pltMaxConn);

	thrust::fill(pltInfoVecs.sumForcesOnPlt.begin(), pltInfoVecs.sumForcesOnPlt.end(), 0);


	thrust::copy(hostPltPosX.begin(), hostPltPosX.end(), pltInfoVecs.pltLocX.begin());
	thrust::copy(hostPltPosY.begin(), hostPltPosY.end(), pltInfoVecs.pltLocY.begin());
	thrust::copy(hostPltPosZ.begin(), hostPltPosZ.end(), pltInfoVecs.pltLocZ.begin());


	std::cout<<"num platelets: "<< pltInfoVecs.pltLocX.size() << std::endl;
	std::cout<<"num platelets var: "<< generalParams.maxPltCount << std::endl;
	//copy fixed positions
	pltInfoVecs.isPltFixed.resize(generalParams.maxPltCount);
	thrust::fill(pltInfoVecs.isPltFixed.begin(), pltInfoVecs.isPltFixed.end(), false);
	//thrust::copy(hostIsPltFixed.begin(), hostIsPltFixed.end(), pltInfoVecs.isPltFixed.begin());


	auxVecs.idPlt_bucket.resize(generalParams.maxPltCount);
	auxVecs.idPlt_value.resize(generalParams.maxPltCount);
	auxVecs.idPlt_bucket_expanded.resize(27 *( generalParams.maxPltCount ));
	auxVecs.idPlt_value_expanded.resize(27 * (generalParams.maxPltCount));

	if (generalParams.currentTime==0.0){
    pltInfoVecs.tndrlNodeId.resize(generalParams.maxPltCount * generalParams.pltMaxConn);
		pltInfoVecs.tndrlNodeType.resize(generalParams.maxPltCount * generalParams.pltMaxConn);
  }

};

void NodeSystemDevice::setTorsionVecs(
	thrust::host_vector<unsigned>& hostTorsionIndexLeft,
	thrust::host_vector<unsigned>& hostTorsionIndexCenter,
	thrust::host_vector<unsigned>& hostTorsionIndexRight,
	thrust::host_vector<double>& hostTorsionAngleZero) {


	torsionInfoVecs.leftIndex.resize(generalParams.totalTorsionCount);
	torsionInfoVecs.centerIndex.resize(generalParams.totalTorsionCount);
	torsionInfoVecs.rightIndex.resize(generalParams.totalTorsionCount);
	torsionInfoVecs.angleZero.resize(generalParams.totalTorsionCount);

	thrust::copy(hostTorsionIndexLeft.begin(), hostTorsionIndexLeft.end(), torsionInfoVecs.leftIndex.begin());
	thrust::copy(hostTorsionIndexCenter.begin(), hostTorsionIndexCenter.end(), torsionInfoVecs.centerIndex.begin());
	thrust::copy(hostTorsionIndexRight.begin(), hostTorsionIndexRight.end(), torsionInfoVecs.rightIndex.begin());

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				torsionInfoVecs.leftIndex.begin(),
				torsionInfoVecs.centerIndex.begin(),
				torsionInfoVecs.rightIndex.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				torsionInfoVecs.leftIndex.begin(),
				torsionInfoVecs.centerIndex.begin(),
				torsionInfoVecs.rightIndex.begin())) + generalParams.totalTorsionCount,
			torsionInfoVecs.angleZero.begin(),//save vector
		TorsionAngleFunctor(
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data())));

	torsionInfoVecs.forceX.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
	torsionInfoVecs.forceY.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
	torsionInfoVecs.forceZ.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
	torsionInfoVecs.tempForceX.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
	torsionInfoVecs.tempForceY.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
	torsionInfoVecs.tempForceZ.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);

	thrust::fill(torsionInfoVecs.forceX.begin(), torsionInfoVecs.forceX.end(), 0.0);
	thrust::fill(torsionInfoVecs.forceY.begin(), torsionInfoVecs.forceY.end(), 0.0);
	thrust::fill(torsionInfoVecs.forceZ.begin(), torsionInfoVecs.forceZ.end(), 0.0);

	torsionInfoVecs.tempTorIndices.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
	torsionInfoVecs.reducedIds.resize(torsionInfoVecs.factor * generalParams.totalTorsionCount);
};

void NodeSystemDevice::setWLCVecs(
	thrust::host_vector<unsigned>& hostWLCSubEdgeLeft,
	thrust::host_vector<unsigned>& hostWLCSubEdgeRight,
	thrust::host_vector<double>& hostWLCSubLenZero ) {

	wlcInfoVecs.globalNeighbors.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);
	wlcInfoVecs.currentNodeEdgeCountVector.resize(generalParams.maxNodeCount);

	wlcInfoVecs.lengthZero.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);
	wlcInfoVecs.numOriginalNeighborsNodeVector.resize(generalParams.maxNodeCount);

	//default value is maxNodeCount
	thrust::fill(wlcInfoVecs.globalNeighbors.begin(), wlcInfoVecs.globalNeighbors.end(), generalParams.maxNodeCount);
	thrust::fill(wlcInfoVecs.currentNodeEdgeCountVector.begin(), wlcInfoVecs.currentNodeEdgeCountVector.end(),0);
	thrust::fill(wlcInfoVecs.lengthZero.begin(), wlcInfoVecs.lengthZero.end(), 0.0);



	nodeInfoVecs.deviceEdgeLeft = hostWLCSubEdgeLeft;
	nodeInfoVecs.deviceEdgeRight = hostWLCSubEdgeRight;

	//scan through hostAdj and put in device.
	for (unsigned id = 0; id < hostWLCSubLenZero.size(); id++) {

		unsigned idL = hostWLCSubEdgeLeft[id];
		unsigned idR = hostWLCSubEdgeRight[id];

		double edgeLen = hostWLCSubLenZero[id];
		//we use the lengthZero vector to identify edges as well.
		//node id is row, column node is connected to row node.

		//add edge for left node
		unsigned edgeNumL = wlcInfoVecs.currentNodeEdgeCountVector[idL]; //number of edges on (nodeId = row)	is that entry in cECV
		unsigned indexL = idL*generalParams.maxNeighborCount + edgeNumL;
		wlcInfoVecs.lengthZero[indexL] = edgeLen;
		wlcInfoVecs.globalNeighbors[indexL] = idR;
		(wlcInfoVecs.currentNodeEdgeCountVector[idL])++; //right connects to left

		//add edge for right node
		unsigned edgeNumR = wlcInfoVecs.currentNodeEdgeCountVector[idR]; //number of edges on (nodeId = row)	is that entry in cECV
		unsigned indexR = idR*generalParams.maxNeighborCount + edgeNumR;
		wlcInfoVecs.lengthZero[indexR] = edgeLen;
		wlcInfoVecs.globalNeighbors[indexR] = idL;
		(wlcInfoVecs.currentNodeEdgeCountVector[idR])++; //left connects to right

		generalParams.currentEdgeCount++;

	}

	//at this point currentNodeEdgeCountVector holds the number of edges, copy this to
	thrust::copy(wlcInfoVecs.currentNodeEdgeCountVector.begin(), wlcInfoVecs.currentNodeEdgeCountVector.end(), wlcInfoVecs.numOriginalNeighborsNodeVector.begin());
};
