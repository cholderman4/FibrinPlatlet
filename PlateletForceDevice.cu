//look up forward declaration and inheritance if this doesn't make sense.
#include "PlateletForceDevice.h"
#include "NodeSystemDevice.h"


typedef thrust::tuple<unsigned, bool> Tub; //tuple holding id of a node and if that node is within reach of the plt

// __host__ __device__

void PltForceOnDevice(
  NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,
	GeneralParams& generalParams,
  PltInfoVecs& pltInfoVecs,
  AuxVecs& auxVecs) {

        thrust::counting_iterator<unsigned> pltindexBegin(0);
        thrust::counting_iterator<unsigned> pltindexEnd(generalParams.maxPltCount);

        //Call the plt force on nodes functor
        thrust::transform(
         thrust::make_zip_iterator(
        	 thrust::make_tuple(
   					auxVecs.bucketPltKeys.begin(),
   					auxVecs.bucketPltValues.begin(),
        		 pltInfoVecs.pltLocX.begin(),
        		 pltInfoVecs.pltLocY.begin(),
        		 pltInfoVecs.pltLocZ.begin())),
         thrust::make_zip_iterator(
        	 thrust::make_tuple(
             auxVecs.bucketPltKeys.begin(),
    				auxVecs.bucketPltValues.begin(),
        		 pltInfoVecs.pltLocX.begin(),
        		 pltInfoVecs.pltLocY.begin(),
        		 pltInfoVecs.pltLocZ.begin())) + generalParams.maxPltCount,
         //save plt forces
         thrust::make_zip_iterator(
        	 thrust::make_tuple(
        		 pltInfoVecs.pltForceX.begin(),
        		 pltInfoVecs.pltForceY.begin(),
        		 pltInfoVecs.pltForceZ.begin())),
//platelets interact with nodes
             PltonNodeForceFunctor(
                 generalParams.pltMaxConn,
                 generalParams.pltRForce,
                 generalParams.pltForce,
                 generalParams.pltR,
                 generalParams.maxPltCount,
                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceX.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceY.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceZ.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedId.data()),
                 thrust::raw_pointer_cast(auxVecs.bucketValuesIncludingNeighbor.data()),
                 thrust::raw_pointer_cast(auxVecs.keyBegin.data()),
                 thrust::raw_pointer_cast(auxVecs.keyEnd.data())
               ));

        //now call a sort by key followed by a reduce by key to figure out which nodes are have force applied.
        //then make a functor that takes the id and force (4 tuple) and takes that force and adds it to the id'th entry in nodeInfoVecs.nodeForceX,Y,Z
        thrust::sort_by_key(pltInfoVecs.nodeUnreducedId.begin(), pltInfoVecs.nodeUnreducedId.begin() + generalParams.pltMaxConn*generalParams.maxPltCount,
        			thrust::make_zip_iterator(
        				thrust::make_tuple(
        					pltInfoVecs.nodeUnreducedForceX.begin(),
        					pltInfoVecs.nodeUnreducedForceY.begin(),
        					pltInfoVecs.nodeUnreducedForceZ.begin())), thrust::less<unsigned>());


        		thrust::fill(pltInfoVecs.nodeReducedForceX.begin(), pltInfoVecs.nodeReducedForceX.end(), 0);
        		thrust::fill(pltInfoVecs.nodeReducedForceY.begin(), pltInfoVecs.nodeReducedForceY.end(), 0);
        		thrust::fill(pltInfoVecs.nodeReducedForceZ.begin(), pltInfoVecs.nodeReducedForceZ.end(), 0);
        		thrust::fill(pltInfoVecs.nodeReducedId.begin(), pltInfoVecs.nodeReducedId.end(), 0);

        		unsigned endKey = thrust::get<0>(
        			thrust::reduce_by_key(
        				pltInfoVecs.nodeUnreducedId.begin(),
        				pltInfoVecs.nodeUnreducedId.begin() + generalParams.pltMaxConn*generalParams.maxPltCount,
        			thrust::make_zip_iterator(
        				thrust::make_tuple(
        					pltInfoVecs.nodeUnreducedForceX.begin(),
        					pltInfoVecs.nodeUnreducedForceY.begin(),
        					pltInfoVecs.nodeUnreducedForceZ.begin())),
        			pltInfoVecs.nodeReducedId.begin(),
        			thrust::make_zip_iterator(
        				thrust::make_tuple(
        					pltInfoVecs.nodeReducedForceX.begin(),
        					pltInfoVecs.nodeReducedForceY.begin(),
        					pltInfoVecs.nodeReducedForceZ.begin())),
        			thrust::equal_to<unsigned>(), CVec3Add())) - pltInfoVecs.nodeReducedId.begin();//binary_pred, binary_op


        		thrust::for_each(
        			thrust::make_zip_iterator(//1st begin
        				thrust::make_tuple(
        					pltInfoVecs.nodeReducedId.begin(),
        					pltInfoVecs.nodeReducedForceX.begin(),
        					pltInfoVecs.nodeReducedForceY.begin(),
        					pltInfoVecs.nodeReducedForceZ.begin())),
        			thrust::make_zip_iterator(//1st end
        				thrust::make_tuple(
        					pltInfoVecs.nodeReducedId.begin(),
        					pltInfoVecs.nodeReducedForceX.begin(),
        					pltInfoVecs.nodeReducedForceY.begin(),
        					pltInfoVecs.nodeReducedForceZ.begin())) + endKey,
        			AddPltonNodeForceFunctor(
        				thrust::raw_pointer_cast(nodeInfoVecs.nodeForceX.data()),
        				thrust::raw_pointer_cast(nodeInfoVecs.nodeForceY.data()),
        				thrust::raw_pointer_cast(nodeInfoVecs.nodeForceZ.data())));

        	}



    // void AdvancePltPosition(//stuff here) {
    //     //stuff here
    // };
