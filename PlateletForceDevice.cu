#include "PlateletForceDevice.h"
#include "NodeSystemDevice.h"

void PltForceOnDevice(
  	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,
	GeneralParams& generalParams,
	PltInfoVecs& pltInfoVecs,
	AuxVecs& auxVecs) {


		thrust::fill(pltInfoVecs.nodeUnreducedForceX.begin(), pltInfoVecs.nodeUnreducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceY.begin(), pltInfoVecs.nodeUnreducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceZ.begin(), pltInfoVecs.nodeUnreducedForceZ.end(), 0.0);

		thrust::fill(pltInfoVecs.nodeReducedForceX.begin(), pltInfoVecs.nodeReducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceY.begin(), pltInfoVecs.nodeReducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceZ.begin(), pltInfoVecs.nodeReducedForceZ.end(), 0.0);

	/*	unsigned begin = auxVecs.keyBegin[auxVecs.bucketPltKeys[0]];
		unsigned end = auxVecs.keyEnd[auxVecs.bucketPltKeys[0]];
		for (unsigned i = begin; i < end; i++) {
			unsigned id = auxVecs.bucketValuesIncludingNeighbor[i];
			std::cout<<id<< std::endl;
		}*/

        //Call the plt force on nodes functor
        thrust::transform(
        	thrust::make_zip_iterator(
        		thrust::make_tuple(
   					auxVecs.bucketPltKeys.begin(),
   					auxVecs.bucketPltValues.begin(),
        			pltInfoVecs.pltLocX.begin(),
        			pltInfoVecs.pltLocY.begin(),
        			pltInfoVecs.pltLocZ.begin())),
        	thrust::make_zip_iterator(
        		thrust::make_tuple(
        			auxVecs.bucketPltKeys.begin(),
    				auxVecs.bucketPltValues.begin(),
        		 	pltInfoVecs.pltLocX.begin(),
        		 	pltInfoVecs.pltLocY.begin(),
        		 	pltInfoVecs.pltLocZ.begin())) + generalParams.maxPltCount,
         //save plt forces
         thrust::make_zip_iterator(
        	 thrust::make_tuple(
				 //reset's forces
        		 pltInfoVecs.pltForceX.begin(),
        		 pltInfoVecs.pltForceY.begin(),
        		 pltInfoVecs.pltForceZ.begin())),
             PltonNodeForceFunctor( 
                 generalParams.pltMaxConn,
                 generalParams.pltRForce,
                 generalParams.pltForce,
                 generalParams.pltR,
                 generalParams.maxPltCount,
                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceX.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceY.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceZ.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedId.data()),
                 thrust::raw_pointer_cast(auxVecs.bucketValuesIncludingNeighbor.data()),
                 thrust::raw_pointer_cast(auxVecs.keyBegin.data()),
                 thrust::raw_pointer_cast(auxVecs.keyEnd.data()) ) ); 

        //now call a sort by key followed by a reduce by key to figure out which nodes are have force applied.
        //then make a functor that takes the id and force (4 tuple) and takes that force and adds it to the id'th entry in nodeInfoVecs.nodeForceX,Y,Z
        thrust::sort_by_key(pltInfoVecs.nodeUnreducedId.begin(), pltInfoVecs.nodeUnreducedId.end(),
        			thrust::make_zip_iterator(
        				thrust::make_tuple(
        					pltInfoVecs.nodeUnreducedForceX.begin(),
        					pltInfoVecs.nodeUnreducedForceY.begin(),
        					pltInfoVecs.nodeUnreducedForceZ.begin())), thrust::less<unsigned>());


        		thrust::fill(pltInfoVecs.nodeReducedForceX.begin(), pltInfoVecs.nodeReducedForceX.end(), 0);
        		thrust::fill(pltInfoVecs.nodeReducedForceY.begin(), pltInfoVecs.nodeReducedForceY.end(), 0);
        		thrust::fill(pltInfoVecs.nodeReducedForceZ.begin(), pltInfoVecs.nodeReducedForceZ.end(), 0);
        		thrust::fill(pltInfoVecs.nodeReducedId.begin(), pltInfoVecs.nodeReducedId.end(), 0);

        		unsigned endKey = thrust::get<0>(
        			thrust::reduce_by_key(
        				pltInfoVecs.nodeUnreducedId.begin(),
        				pltInfoVecs.nodeUnreducedId.end(),
        			thrust::make_zip_iterator(
        				thrust::make_tuple(
        					pltInfoVecs.nodeUnreducedForceX.begin(),
        					pltInfoVecs.nodeUnreducedForceY.begin(),
        					pltInfoVecs.nodeUnreducedForceZ.begin())),
        			pltInfoVecs.nodeReducedId.begin(),
        			thrust::make_zip_iterator(
        				thrust::make_tuple(
        					pltInfoVecs.nodeReducedForceX.begin(),
        					pltInfoVecs.nodeReducedForceY.begin(),
        					pltInfoVecs.nodeReducedForceZ.begin())),
        			thrust::equal_to<unsigned>(), CVec3Add())) - pltInfoVecs.nodeReducedId.begin();//binary_pred, binary_op


        		thrust::for_each(
        			thrust::make_zip_iterator(//1st begin
        				thrust::make_tuple(
        					pltInfoVecs.nodeReducedId.begin(),
        					pltInfoVecs.nodeReducedForceX.begin(),
        					pltInfoVecs.nodeReducedForceY.begin(),
        					pltInfoVecs.nodeReducedForceZ.begin())),
        			thrust::make_zip_iterator(//1st end
        				thrust::make_tuple(
        					pltInfoVecs.nodeReducedId.begin(),
        					pltInfoVecs.nodeReducedForceX.begin(),
        					pltInfoVecs.nodeReducedForceY.begin(),
        					pltInfoVecs.nodeReducedForceZ.begin())) + endKey,
        			AddPltonNodeForceFunctor(
        				thrust::raw_pointer_cast(nodeInfoVecs.nodeForceX.data()),
        				thrust::raw_pointer_cast(nodeInfoVecs.nodeForceY.data()),
        				thrust::raw_pointer_cast(nodeInfoVecs.nodeForceZ.data())));

        	}



    // void AdvancePltPosition(//stuff here) {
    //     //stuff here
    // };
