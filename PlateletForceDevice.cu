#include "PlateletForceDevice.h"
#include "NodeSystemDevice.h"

//Force field-like mode
void PltForceOnDevice(
  	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,
	GeneralParams& generalParams,
	PltInfoVecs& pltInfoVecs,
	AuxVecs& auxVecs) {


		thrust::fill(pltInfoVecs.nodeUnreducedForceX.begin(), pltInfoVecs.nodeUnreducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceY.begin(), pltInfoVecs.nodeUnreducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceZ.begin(), pltInfoVecs.nodeUnreducedForceZ.end(), 0.0);

		thrust::fill(pltInfoVecs.nodeReducedForceX.begin(), pltInfoVecs.nodeReducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceY.begin(), pltInfoVecs.nodeReducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceZ.begin(), pltInfoVecs.nodeReducedForceZ.end(), 0.0);

		//fill for image sort
    	thrust::fill(pltInfoVecs.nodeUnreducedId.begin(),pltInfoVecs.nodeUnreducedId.end(), generalParams.maxNodeCount);
	/*	unsigned begin = auxVecs.keyBegin[auxVecs.idPlt_bucket[0]];
		unsigned end = auxVecs.keyEnd[auxVecs.idPlt_bucket[0]];
		for (unsigned i = begin; i < end; i++) {
			unsigned id = auxVecs.id_bucket_expanded[i];
			std::cout<<id<< std::endl;
		}*/

        //Call the plt force on nodes functor
        thrust::transform(
        	thrust::make_zip_iterator(
        		thrust::make_tuple(
   					auxVecs.idPlt_bucket.begin(),
   					auxVecs.idPlt_value.begin(),
        			pltInfoVecs.pltLocX.begin(),
        			pltInfoVecs.pltLocY.begin(),
        			pltInfoVecs.pltLocZ.begin())),
        	thrust::make_zip_iterator(
        		thrust::make_tuple(
        			auxVecs.idPlt_bucket.begin(),
    				auxVecs.idPlt_value.begin(),
        		 	pltInfoVecs.pltLocX.begin(),
        		 	pltInfoVecs.pltLocY.begin(),
        		 	pltInfoVecs.pltLocZ.begin())) + generalParams.maxPltCount,
         //save plt forces
         thrust::make_zip_iterator(
        	 thrust::make_tuple(
				 //reset's forces
        		 pltInfoVecs.pltForceX.begin(),
        		 pltInfoVecs.pltForceY.begin(),
        		 pltInfoVecs.pltForceZ.begin())),
             PltonNodeForceFunctor(
                 generalParams.pltMaxConn,
                 generalParams.pltRForce,
                 generalParams.pltForce,
                 generalParams.pltR,
                 generalParams.maxPltCount,
                 generalParams.fiberDiameter,
				 generalParams.maxNodeCount,

                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceX.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceY.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceZ.data()),

                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedId.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.pltImagingConnection.data()),

                 thrust::raw_pointer_cast(auxVecs.id_bucket_expanded.data()),
                 thrust::raw_pointer_cast(auxVecs.keyBegin.data()),
                 thrust::raw_pointer_cast(auxVecs.keyEnd.data()) ) );

        //now call a sort by key followed by a reduce by key to figure out which nodes are have force applied.
        //then make a functor that takes the id and force (4 tuple) and takes that force and adds it to the id'th entry in nodeInfoVecs.nodeForceX,Y,Z
        thrust::sort_by_key(pltInfoVecs.nodeUnreducedId.begin(), pltInfoVecs.nodeUnreducedId.end(),
        			thrust::make_zip_iterator(
        				thrust::make_tuple(
							pltInfoVecs.pltImagingConnection.begin(),
        					pltInfoVecs.nodeUnreducedForceX.begin(),
        					pltInfoVecs.nodeUnreducedForceY.begin(),
        					pltInfoVecs.nodeUnreducedForceZ.begin())), thrust::less<unsigned>());

    thrust::copy(pltInfoVecs.nodeUnreducedId.begin(),pltInfoVecs.nodeUnreducedId.end(), pltInfoVecs.nodeImagingConnection.begin());

    pltInfoVecs.numConnections = thrust::count_if(
        pltInfoVecs.nodeImagingConnection.begin(),
        pltInfoVecs.nodeImagingConnection.end(), is_less_than(generalParams.maxNodeCount) );

	//std::cout<<pltInfoVecs.numConnections<<std::endl;
	/*for (unsigned i = 0; i < pltInfoVecs.nodeImagingConnection.size(); i ++) {
		std::cout<<pltInfoVecs.nodeImagingConnection[i]<<std::endl;
	}*/

	//std::cout<<"num connect: "<< pltInfoVecs.numConnections<<std::endl;
	//std::cout<<"num nod imaging: "<< pltInfoVecs.nodeImagingConnection.size() <<std::endl;
	//std::cout<<"num node unreduced: "<< pltInfoVecs.nodeUnreducedId.size() <<std::endl;
	//std::cout<<"num plt imaging: "<< pltInfoVecs.pltImagingConnection.size() <<std::endl;

	/*for (unsigned i = 0; i < pltInfoVecs.nodeUnreducedId.size(); i ++) {
		std::cout<<pltInfoVecs.nodeUnreducedId[i]<<std::endl;
		std::cout<<pltInfoVecs.nodeUnreducedForceX[i]<<std::endl;
		std::cout<<pltInfoVecs.nodeUnreducedForceY[i]<<std::endl;
		std::cout<<pltInfoVecs.nodeUnreducedForceZ[i]<<std::endl;
	}*/

/*	for (unsigned i = 0; i < nodeInfoVecs.nodeLocX.size(); i ++) {
		std::cout<<"node Loc "<< nodeInfoVecs.nodeLocX[i] << " " << nodeInfoVecs.nodeLocY[i] <<" "<< nodeInfoVecs.nodeLocZ[i] << std::endl;

	}*/
//	std::cout<<"plt Loc "<< pltInfoVecs.pltLocX[0] << " " << pltInfoVecs.pltLocY[0] <<" "<< pltInfoVecs.pltLocZ[0] << std::endl;
//	std::cout<<"plt Loc "<< pltInfoVecs.pltLocX[1] << " " << pltInfoVecs.pltLocY[1] <<" "<< pltInfoVecs.pltLocZ[1] << std::endl;


//reduce and apply force
 		unsigned endKey = thrust::get<0>(
 			thrust::reduce_by_key(
 				pltInfoVecs.nodeUnreducedId.begin(),
 				pltInfoVecs.nodeUnreducedId.end(),
 			thrust::make_zip_iterator(
 				thrust::make_tuple(
 					pltInfoVecs.nodeUnreducedForceX.begin(),
 					pltInfoVecs.nodeUnreducedForceY.begin(),
 					pltInfoVecs.nodeUnreducedForceZ.begin())),
 			pltInfoVecs.nodeReducedId.begin(),
 			thrust::make_zip_iterator(
 				thrust::make_tuple(//need t check
 					pltInfoVecs.nodeReducedForceX.begin(),
 					pltInfoVecs.nodeReducedForceY.begin(),
 					pltInfoVecs.nodeReducedForceZ.begin())),
 			thrust::equal_to<unsigned>(), CVec3Add())) - pltInfoVecs.nodeReducedId.begin();//binary_pred, binary_op


        thrust::for_each(
        	thrust::make_zip_iterator(//1st begin
        		thrust::make_tuple(
        			pltInfoVecs.nodeReducedId.begin(),
        			pltInfoVecs.nodeReducedForceX.begin(),
        			pltInfoVecs.nodeReducedForceY.begin(),
        			pltInfoVecs.nodeReducedForceZ.begin())),
        	thrust::make_zip_iterator(//1st end
        		thrust::make_tuple(
        			pltInfoVecs.nodeReducedId.begin(),
        			pltInfoVecs.nodeReducedForceX.begin(),
        			pltInfoVecs.nodeReducedForceY.begin(),
        			pltInfoVecs.nodeReducedForceZ.begin())) + endKey,
        	AddPltonNodeForceFunctor(
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceX.data()),
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceY.data()),
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceZ.data())));
};

void PltInteractionOnDevice(
  	GeneralParams& generalParams,
  	PltInfoVecs& pltInfoVecs,
  	AuxVecs& auxVecs) {

    thrust::for_each(
      	thrust::make_zip_iterator(
        	thrust::make_tuple(
        		auxVecs.idPlt_bucket.begin(),
        		auxVecs.idPlt_value.begin(),
          		pltInfoVecs.pltLocX.begin(),
          		pltInfoVecs.pltLocY.begin(),
          		pltInfoVecs.pltLocZ.begin())),
    thrust::make_zip_iterator(
        thrust::make_tuple(
          		auxVecs.idPlt_bucket.begin(),
        		auxVecs.idPlt_value.begin(),
          		pltInfoVecs.pltLocX.begin(),
          		pltInfoVecs.pltLocY.begin(),
          		pltInfoVecs.pltLocZ.begin())) + generalParams.maxPltCount,

         PltonPltForceFunctor(
             generalParams.pltMaxConn,
             generalParams.pltRForce,
             generalParams.pltForce,
             generalParams.pltR,
             generalParams.maxPltCount,
             thrust::raw_pointer_cast(pltInfoVecs.pltLocX.data()),
             thrust::raw_pointer_cast(pltInfoVecs.pltLocY.data()),
             thrust::raw_pointer_cast(pltInfoVecs.pltLocZ.data()),
             thrust::raw_pointer_cast(pltInfoVecs.pltForceX.data()),
             thrust::raw_pointer_cast(pltInfoVecs.pltForceY.data()),
             thrust::raw_pointer_cast(pltInfoVecs.pltForceZ.data()),
             thrust::raw_pointer_cast(auxVecs.idPlt_value_expanded.data()),
             thrust::raw_pointer_cast(auxVecs.keyPltBegin.data()),
             thrust::raw_pointer_cast(auxVecs.keyPltEnd.data()) ) );
};



//tendril-like force
void PltTndrlOnDevice(
  NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,
	GeneralParams& generalParams,
	PltInfoVecs& pltInfoVecs,
	AuxVecs& auxVecs) {

  if (generalParams.currentTime==0.0){
    thrust::fill(pltInfoVecs.tndrlNodeId.begin(),pltInfoVecs.tndrlNodeId.end(), generalParams.maxNodeCount + generalParams.maxPltCount);
    }
		thrust::fill(pltInfoVecs.nodeUnreducedForceX.begin(), pltInfoVecs.nodeUnreducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceY.begin(), pltInfoVecs.nodeUnreducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceZ.begin(), pltInfoVecs.nodeUnreducedForceZ.end(), 0.0);

		thrust::fill(pltInfoVecs.nodeReducedForceX.begin(), pltInfoVecs.nodeReducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceY.begin(), pltInfoVecs.nodeReducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceZ.begin(), pltInfoVecs.nodeReducedForceZ.end(), 0.0);

		//fill for image sort
    	thrust::fill(pltInfoVecs.nodeUnreducedId.begin(),pltInfoVecs.nodeUnreducedId.end(), generalParams.maxNodeCount);
	/*	unsigned begin = auxVecs.keyBegin[auxVecs.idPlt_bucket[0]];
		unsigned end = auxVecs.keyEnd[auxVecs.idPlt_bucket[0]];
		for (unsigned i = begin; i < end; i++) {
			unsigned id = auxVecs.id_bucket_expanded[i];
			std::cout<<id<< std::endl;
		}*/

        //Call the plt force on nodes functor
        thrust::transform(
        	thrust::make_zip_iterator(
        		thrust::make_tuple(
   					auxVecs.idPlt_bucket.begin(),
   					auxVecs.idPlt_value.begin(),
        			pltInfoVecs.pltLocX.begin(),
        			pltInfoVecs.pltLocY.begin(),
        			pltInfoVecs.pltLocZ.begin())),
        	thrust::make_zip_iterator(
        		thrust::make_tuple(
        			auxVecs.idPlt_bucket.begin(),
    				auxVecs.idPlt_value.begin(),
        		 	pltInfoVecs.pltLocX.begin(),
        		 	pltInfoVecs.pltLocY.begin(),
        		 	pltInfoVecs.pltLocZ.begin())) + generalParams.maxPltCount,
         //save plt forces
         thrust::make_zip_iterator(
        	 thrust::make_tuple(
				 //reset's forces
        		 pltInfoVecs.pltForceX.begin(),
        		 pltInfoVecs.pltForceY.begin(),
        		 pltInfoVecs.pltForceZ.begin())),
             PltTndrlonNodeForceFunctor(
                 generalParams.pltMaxConn,
                 generalParams.pltRForce,
                 generalParams.pltForce,
                 generalParams.pltR,
                 generalParams.maxPltCount,
                 generalParams.fiberDiameter,
		             generalParams.maxNodeCount,
                 generalParams.maxNeighborCount,

                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceX.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceY.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceZ.data()),

                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedId.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.pltImagingConnection.data()),

                 thrust::raw_pointer_cast(auxVecs.id_bucket_expanded.data()),
                 thrust::raw_pointer_cast(auxVecs.keyBegin.data()),
                 thrust::raw_pointer_cast(auxVecs.keyEnd.data()),

                 thrust::raw_pointer_cast(pltInfoVecs.tndrlNodeId.data()),
                 thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data())
                 thrust::raw_pointer_cast(nodeInfoVecs.pltLocX.data()),
                 thrust::raw_pointer_cast(nodeInfoVecs.pltLocY.data()),
                 thrust::raw_pointer_cast(nodeInfoVecs.pltLocZ.data()),) );

        //now call a sort by key followed by a reduce by key to figure out which nodes are have force applied.
        //then make a functor that takes the id and force (4 tuple) and takes that force and adds it to the id'th entry in nodeInfoVecs.nodeForceX,Y,Z
        thrust::sort_by_key(pltInfoVecs.nodeUnreducedId.begin(), pltInfoVecs.nodeUnreducedId.end(),
        			thrust::make_zip_iterator(
        				thrust::make_tuple(
							pltInfoVecs.pltImagingConnection.begin(),
        					pltInfoVecs.nodeUnreducedForceX.begin(),
        					pltInfoVecs.nodeUnreducedForceY.begin(),
        					pltInfoVecs.nodeUnreducedForceZ.begin())), thrust::less<unsigned>());

    thrust::copy(pltInfoVecs.nodeUnreducedId.begin(),pltInfoVecs.nodeUnreducedId.end(), pltInfoVecs.nodeImagingConnection.begin());

    pltInfoVecs.numConnections = thrust::count_if(
        pltInfoVecs.nodeImagingConnection.begin(),
        pltInfoVecs.nodeImagingConnection.end(), is_less_than(generalParams.maxNodeCount) );

	//std::cout<<pltInfoVecs.numConnections<<std::endl;
	/*for (unsigned i = 0; i < pltInfoVecs.nodeImagingConnection.size(); i ++) {
		std::cout<<pltInfoVecs.nodeImagingConnection[i]<<std::endl;
	}*/

	//std::cout<<"num connect: "<< pltInfoVecs.numConnections<<std::endl;
	//std::cout<<"num nod imaging: "<< pltInfoVecs.nodeImagingConnection.size() <<std::endl;
	//std::cout<<"num node unreduced: "<< pltInfoVecs.nodeUnreducedId.size() <<std::endl;
	//std::cout<<"num plt imaging: "<< pltInfoVecs.pltImagingConnection.size() <<std::endl;

	/*for (unsigned i = 0; i < pltInfoVecs.nodeUnreducedId.size(); i ++) {
		std::cout<<pltInfoVecs.nodeUnreducedId[i]<<std::endl;
		std::cout<<pltInfoVecs.nodeUnreducedForceX[i]<<std::endl;
		std::cout<<pltInfoVecs.nodeUnreducedForceY[i]<<std::endl;
		std::cout<<pltInfoVecs.nodeUnreducedForceZ[i]<<std::endl;
	}*/

/*	for (unsigned i = 0; i < nodeInfoVecs.nodeLocX.size(); i ++) {
		std::cout<<"node Loc "<< nodeInfoVecs.nodeLocX[i] << " " << nodeInfoVecs.nodeLocY[i] <<" "<< nodeInfoVecs.nodeLocZ[i] << std::endl;

	}*/
//	std::cout<<"plt Loc "<< pltInfoVecs.pltLocX[0] << " " << pltInfoVecs.pltLocY[0] <<" "<< pltInfoVecs.pltLocZ[0] << std::endl;
//	std::cout<<"plt Loc "<< pltInfoVecs.pltLocX[1] << " " << pltInfoVecs.pltLocY[1] <<" "<< pltInfoVecs.pltLocZ[1] << std::endl;


//reduce and apply force
 		unsigned endKey = thrust::get<0>(
 			thrust::reduce_by_key(
 				pltInfoVecs.nodeUnreducedId.begin(),
 				pltInfoVecs.nodeUnreducedId.end(),
 			thrust::make_zip_iterator(
 				thrust::make_tuple(
 					pltInfoVecs.nodeUnreducedForceX.begin(),
 					pltInfoVecs.nodeUnreducedForceY.begin(),
 					pltInfoVecs.nodeUnreducedForceZ.begin())),
 			pltInfoVecs.nodeReducedId.begin(),
 			thrust::make_zip_iterator(
 				thrust::make_tuple(//need t check
 					pltInfoVecs.nodeReducedForceX.begin(),
 					pltInfoVecs.nodeReducedForceY.begin(),
 					pltInfoVecs.nodeReducedForceZ.begin())),
 			thrust::equal_to<unsigned>(), CVec3Add())) - pltInfoVecs.nodeReducedId.begin();//binary_pred, binary_op


        thrust::for_each(
        	thrust::make_zip_iterator(//1st begin
        		thrust::make_tuple(
        			pltInfoVecs.nodeReducedId.begin(),
        			pltInfoVecs.nodeReducedForceX.begin(),
        			pltInfoVecs.nodeReducedForceY.begin(),
        			pltInfoVecs.nodeReducedForceZ.begin())),
        	thrust::make_zip_iterator(//1st end
        		thrust::make_tuple(
        			pltInfoVecs.nodeReducedId.begin(),
        			pltInfoVecs.nodeReducedForceX.begin(),
        			pltInfoVecs.nodeReducedForceY.begin(),
        			pltInfoVecs.nodeReducedForceZ.begin())) + endKey,
        	AddPltonNodeForceFunctor(
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceX.data()),
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceY.data()),
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceZ.data())));
};

/*void PltTndrlInteractionOnDevice(
  	GeneralParams& generalParams,
  	PltInfoVecs& pltInfoVecs,
  	AuxVecs& auxVecs) {

    thrust::for_each(
      	thrust::make_zip_iterator(
        	thrust::make_tuple(
        		auxVecs.idPlt_bucket.begin(),
        		auxVecs.idPlt_value.begin(),
          		pltInfoVecs.pltLocX.begin(),
          		pltInfoVecs.pltLocY.begin(),
          		pltInfoVecs.pltLocZ.begin())),
    thrust::make_zip_iterator(
        thrust::make_tuple(
          		auxVecs.idPlt_bucket.begin(),
        		auxVecs.idPlt_value.begin(),
          		pltInfoVecs.pltLocX.begin(),
          		pltInfoVecs.pltLocY.begin(),
          		pltInfoVecs.pltLocZ.begin())) + generalParams.maxPltCount,

         PltonPltForceFunctor(
             generalParams.pltMaxConn,
             generalParams.pltRForce,
             generalParams.pltForce,
             generalParams.pltR,
             generalParams.maxPltCount,
             thrust::raw_pointer_cast(pltInfoVecs.pltLocX.data()),
             thrust::raw_pointer_cast(pltInfoVecs.pltLocY.data()),
             thrust::raw_pointer_cast(pltInfoVecs.pltLocZ.data()),
             thrust::raw_pointer_cast(pltInfoVecs.pltForceX.data()),
             thrust::raw_pointer_cast(pltInfoVecs.pltForceY.data()),
             thrust::raw_pointer_cast(pltInfoVecs.pltForceZ.data()),
             thrust::raw_pointer_cast(auxVecs.idPlt_value_expanded.data()),
             thrust::raw_pointer_cast(auxVecs.keyPltBegin.data()),
             thrust::raw_pointer_cast(auxVecs.keyPltEnd.data()) ) );
};*/
