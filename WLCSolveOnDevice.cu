#include "NodeSystemDevice.h"
#include "WLCSolveOnDevice.h" 

void WLCSolveOnDevice(
	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,  
	GeneralParams& generalParams) {
 
 
	thrust::counting_iterator<unsigned> startEdgeIter(0);
			  
	//
	thrust::for_each( 
		thrust::make_zip_iterator( 
			thrust::make_tuple(startEdgeIter,
								nodeInfoVecs.isNodeFixed.begin() )),
		thrust::make_zip_iterator(
			thrust::make_tuple(startEdgeIter,
								nodeInfoVecs.isNodeFixed.begin() )) + generalParams.maxNodeCount,
		WLCfunctor(
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeForceX.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeForceY.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeForceZ.data()),
 
			generalParams.kB,
			generalParams.persistenceLengthMon,
			generalParams.CLM,
			generalParams.temperature,
			generalParams.maxNeighborCount,
			generalParams.maxNodeCount,

			thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()),
			thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
			thrust::raw_pointer_cast(wlcInfoVecs.currentNodeEdgeCountVector.data()),
			thrust::raw_pointer_cast(wlcInfoVecs.numOriginalNeighborsNodeVector.data()) ) );
};

